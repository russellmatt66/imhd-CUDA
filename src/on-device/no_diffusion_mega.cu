#include <iostream>
#include <cstdlib>
#include <cstring>
#include <assert.h>
#include <stdio.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>

#include "initialize_od.cuh"
#include "kernels_od.cuh"
#include "kernels_fluidbcs.cuh"
#include "kernels_od_intvar.cuh"
#include "kernels_intvarbcs.cuh"

#include "utils.cuh"
#include "utils.hpp"

int main(int argc, char* argv[]){
   int Nt = atoi(argv[1]);
   int Nx = atoi(argv[2]);
   int Ny = atoi(argv[3]);
   int Nz = atoi(argv[4]);

   float J0 = atof(argv[5]); // amplitude of the current
	float r_max_coeff = atof(argv[7]); // r_pinch = r_max_coeff * r_max
	
   float x_min = atof(argv[8]);
	float x_max = atof(argv[9]);
	float y_min = atof(argv[10]);
	float y_max = atof(argv[11]);
	float z_min = atof(argv[12]);
	float z_max = atof(argv[13]);
	float dt = atof(argv[14]);
   
   std::string path_to_data = argv[15];
   std::string phdf5_bin_name = argv[16];
   std::string attr_bin_name = argv[17];
   std::string write_grid_bin_name = argv[18];
   std::string eigen_bin_name = argv[19];
   std::string num_proc = argv[20];

   int xgrid_threads = atoi(argv[21]);
   int ygrid_threads = atoi(argv[22]);
   int zgrid_threads = atoi(argv[23]);

   int init_xthreads = atoi(argv[24]);
   int init_ythreads = atoi(argv[25]);
   int init_zthreads = atoi(argv[26]);
   
   int FA_xthreads = atoi(argv[27]);
   int FA_ythreads = atoi(argv[28]);
   int FA_zthreads = atoi(argv[29]);

   int SM_mult_grid_x = atoi(argv[39]);
   int SM_mult_grid_y = atoi(argv[40]);
   int SM_mult_grid_z = atoi(argv[41]);

	int SM_mult_init_x = atoi(argv[42]);
	int SM_mult_init_y = atoi(argv[43]);
	int SM_mult_init_z = atoi(argv[44]);

   int SM_mult_FA_x = atoi(argv[45]);
   int SM_mult_FA_y = atoi(argv[46]);
   int SM_mult_FA_z = atoi(argv[47]);

   // CUDA BOILERPLATE 
   int deviceId;
   int numberOfSMs;

   hipGetDevice(&deviceId); // number of blocks should be a multiple of the number of device SMs
   hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

   size_t cube_size = Nx * Ny * Nz;
   size_t fluidvar_size = sizeof(float) * cube_size;
   size_t fluid_data_size = 8 * fluidvar_size;

   float *fluidvars, *intvars;

   checkCuda(hipMalloc(&fluidvars, fluid_data_size));
   checkCuda(hipMalloc(&intvars, fluid_data_size));
   
   float *x_grid, *y_grid, *z_grid;

   checkCuda(hipMalloc(&x_grid, sizeof(float) * Nx));
   checkCuda(hipMalloc(&y_grid, sizeof(float) * Ny));
   checkCuda(hipMalloc(&z_grid, sizeof(float) * Nz));

   float dx = (x_max - x_min) / (Nx - 1);
	float dy = (y_max - y_min) / (Ny - 1);
	float dz = (z_max - z_min) / (Nz - 1);

   dim3 egd_grid(SM_mult_grid_x * numberOfSMs, SM_mult_grid_y * numberOfSMs, SM_mult_grid_z * numberOfSMs);
   dim3 tbd_grid(xgrid_threads, ygrid_threads, zgrid_threads);

   dim3 egd_init(SM_mult_init_x * numberOfSMs, SM_mult_init_y * numberOfSMs, SM_mult_init_z * numberOfSMs);
   dim3 tbd_init(init_xthreads, init_ythreads, init_zthreads); 
   
   InitializeGrid<<<egd_grid, tbd_grid>>>(x_min, x_max, y_min, y_max, z_min, z_max, dx, dy, dz, x_grid, y_grid, z_grid, Nx, Ny, Nz);
   ZeroVars<<<egd_init, tbd_init>>>(fluidvars, Nx, Ny, Nz);
   checkCuda(hipDeviceSynchronize());

   ScrewPinchStride<<<egd_init, tbd_init>>>(fluidvars, J0, x_grid, y_grid, z_grid, Nx, Ny, Nz); 
   // ScrewPinch<<<egd_init, tbd_init>>>(fluidvars, J0, r_max_coeff, x_grid, y_grid, z_grid, Nx, Ny, Nz); /* TODO: Specify optimized egd+tbd for this */ 
   checkCuda(hipDeviceSynchronize());

   dim3 egd_fluidadvance(SM_mult_FA_x * numberOfSMs, SM_mult_FA_y * numberOfSMs, SM_mult_FA_z * numberOfSMs);
   dim3 tbd_fluidadvance(FA_xthreads, FA_ythreads, FA_zthreads); 

   ComputeIntermediateVariablesNoDiff<<<egd_fluidadvance, tbd_fluidadvance>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
   checkCuda(hipDeviceSynchronize());    

   /* NOTE: Hella slow b/c megakernels */
   ComputeIntermediateVariablesBoundaryNoDiff<<<egd_fluidadvance, tbd_fluidadvance>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz); 
   checkCuda(hipDeviceSynchronize());    

   // Use IPC to write data out in order to avoid redundant work 
   std::string shm_name_fluidvar = "/shared_h_fluidvar";
   int shm_fd = shm_open(shm_name_fluidvar.data(), O_CREAT | O_RDWR, 0666);
   if (shm_fd == -1) {
      std::cerr << "Failed to create shared memory!" << std::endl;
      return EXIT_FAILURE;
   }
   
   ftruncate(shm_fd, fluid_data_size);
   float* shm_h_fluidvar = (float*)mmap(0, fluid_data_size, PROT_READ | PROT_WRITE, MAP_SHARED, shm_fd, 0);
   if (shm_h_fluidvar == MAP_FAILED) {
      std::cerr << "mmap failed!" << std::endl;
      return EXIT_FAILURE;
   }

   std::cout << "Transferring device data to host" << std::endl;
   hipMemcpy(shm_h_fluidvar, fluidvars, fluid_data_size, hipMemcpyDeviceToHost);
   checkCuda(hipDeviceSynchronize());

   std::string filename_fluidvar = path_to_data + "fluidvars_0.h5";

   std::cout << "Writing Screw-Pinch ICs out with PHDF5" << std::endl;
   int ret = callBinary_PHDF5Write(filename_fluidvar, Nx, Ny, Nz, shm_name_fluidvar, fluid_data_size, num_proc, phdf5_bin_name); 
   if (ret != 0) {
        std::cerr << "Error executing PHDF5 command" << std::endl;
   }

   std::cout << "Writing attributes to the dataset with HDF5" << std::endl;
   ret = callBinary_AttrWrite(filename_fluidvar, Nx, Ny, Nz, attr_bin_name); // inadvisable to write attributes in a PHDF5 context
   if (ret != 0) {
        std::cerr << "Error executing attribute command" << std::endl;
   }

   // COMPUTE STABILITY CRITERION
   // First, transfer grid data
   std::string shm_name_gridx = "/shared_h_gridx";
   shm_fd = shm_open(shm_name_gridx.data(), O_CREAT | O_RDWR, 0666);
   ftruncate(shm_fd, sizeof(float) * Nx);
   float* shm_h_gridx = (float*)mmap(0, sizeof(float) * Nx, PROT_READ | PROT_WRITE, MAP_SHARED, shm_fd, 0);
   if (shm_h_gridx == MAP_FAILED) {
      std::cerr << "mmap failed for grid_x!" << std::endl;
      return EXIT_FAILURE;
   }
   hipMemcpy(shm_h_gridx, x_grid, sizeof(float) * Nx, hipMemcpyDeviceToHost);

   std::string shm_name_gridy = "/shared_h_gridy";
   shm_fd = shm_open(shm_name_gridy.data(), O_CREAT | O_RDWR, 0666);
   ftruncate(shm_fd, sizeof(float) * Ny);
   float* shm_h_gridy = (float*)mmap(0, sizeof(float) * Ny, PROT_READ | PROT_WRITE, MAP_SHARED, shm_fd, 0);
   if (shm_h_gridy == MAP_FAILED) {
      std::cerr << "mmap failed for grid_y!" << std::endl;
      return EXIT_FAILURE;
   }
   hipMemcpy(shm_h_gridy, y_grid, sizeof(float) * Ny, hipMemcpyDeviceToHost);

   std::string shm_name_gridz = "/shared_h_gridz";
   shm_fd = shm_open(shm_name_gridz.data(), O_CREAT | O_RDWR, 0666);
   ftruncate(shm_fd, sizeof(float) * Nz);
   float* shm_h_gridz = (float*)mmap(0, sizeof(float) * Nz, PROT_READ | PROT_WRITE, MAP_SHARED, shm_fd, 0);
   if (shm_h_gridz == MAP_FAILED) {
      std::cerr << "mmap failed for grid_z!" << std::endl;
      return EXIT_FAILURE;
   }
   hipMemcpy(shm_h_gridz, z_grid, sizeof(float) * Nz, hipMemcpyDeviceToHost);
   checkCuda(hipDeviceSynchronize());

   std::cout << "Forking to process for writing grid to storage" << std::endl;
   ret = callBinary_WriteGrid(write_grid_bin_name, path_to_data, shm_name_gridx, shm_name_gridy, shm_name_gridz, Nx, Ny, Nz);
   if (ret != 0) {
         std::cerr << "Error executing writegrid binary: " << eigen_bin_name << std::endl;
   }

   if (!(eigen_bin_name == "none")){ // Don't always want to check stability - expensive raster scan
      std::cout << "Forking to process for computing CFL number (checking stability)" << std::endl;
      ret = callBinary_EigenSC(shm_name_fluidvar, Nx, Ny, Nz, eigen_bin_name, dt, dx, dy, dz, shm_name_gridx, shm_name_gridy, shm_name_gridz);
      if (ret != 0) {
         std::cerr << "Error executing Eigen binary: " << eigen_bin_name << std::endl;
         std::cerr << "Error code: " << ret << std::endl;
      }
   }

   // SIMULATION LOOP
   for (int it = 1; it < Nt; it++){
      std::cout << "Starting timestep " << it << std::endl;

      std::cout << "Launching kernel for computing fluid variables" << std::endl;
      FluidAdvanceLocalNoDiff<<<egd_fluidadvance, tbd_fluidadvance>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
      checkCuda(hipDeviceSynchronize());
      
      std::cout << "Launching kernel for computing fluid boundaries" << std::endl; 
      BoundaryConditionsNoDiff<<<egd_fluidadvance, tbd_fluidadvance>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz); // Hella slow b/c megakernel
      checkCuda(hipDeviceSynchronize());
      std::cout << "Kernels for computing fluid variables completed" << std::endl;
      
      std::cout << "Launching kernel for computing intermediate variables" << std::endl; 
      ComputeIntermediateVariablesNoDiff<<<egd_fluidadvance, tbd_fluidadvance>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz); 
      checkCuda(hipDeviceSynchronize());

      std::cout << "Launching kernels for computing intermediate boundaries" << std::endl; 
      ComputeIntermediateVariablesBoundaryNoDiff<<<egd_fluidadvance, tbd_fluidadvance>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz); // Hella slow b/c megakernel
      
      std::cout << "Transferring updated fluid data to host" << std::endl;
      hipMemcpy(shm_h_fluidvar, fluidvars, fluid_data_size, hipMemcpyDeviceToHost);
      checkCuda(hipDeviceSynchronize());
      std::cout << "Kernels for computing intermediate variables completed" << std::endl;
      std::cout << "Fluid D2H data migration completed" << std::endl;

      std::cout << "Writing updated fluid data out" << std::endl;
      filename_fluidvar = path_to_data + "fluidvars_" + std::to_string(it) + ".h5";
      ret = callBinary_PHDF5Write(filename_fluidvar, Nx, Ny, Nz, shm_name_fluidvar, fluid_data_size, num_proc, phdf5_bin_name);
      if (ret != 0) {
         std::cerr << "Error forking PHDF5Write binary on timestep " << std::to_string(it) << std::endl;
      }  

      std::cout << "Timestep " << it << " complete" << std::endl;

      if (!(eigen_bin_name == "none")){ // Don't always want to check stability - expensive raster scan
         std::cout << "Forking to process for computing CFL number (checking stability)" << std::endl;
         ret = callBinary_EigenSC(shm_name_fluidvar, Nx, Ny, Nz, eigen_bin_name, dt, dx, dy, dz, shm_name_gridx, shm_name_gridy, shm_name_gridz);
         if (ret != 0) {
            std::cerr << "Error executing Eigen binary: " << eigen_bin_name << std::endl;
            std::cerr << "Error code: " << ret << std::endl;
         }
      }  
   } 

   // FREE EVERYTHING
   // Device
   checkCuda(hipFree(fluidvars));
   checkCuda(hipFree(intvars));
   checkCuda(hipFree(x_grid));
   checkCuda(hipFree(y_grid));
   checkCuda(hipFree(z_grid));
   
   // Host
   munmap(shm_h_fluidvar, 8 * fluid_data_size);
   munmap(shm_h_gridx, sizeof(float) * Nx);
   munmap(shm_h_gridy, sizeof(float) * Ny);
   munmap(shm_h_gridz, sizeof(float) * Nz);
   shm_unlink(shm_name_fluidvar.data());
   shm_unlink(shm_name_gridx.data());
   shm_unlink(shm_name_gridy.data());
   shm_unlink(shm_name_gridz.data());
   return 0;
}