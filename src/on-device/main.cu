#include <iostream>
#include <cstdlib>
#include <cstring>
#include <assert.h>
#include <stdio.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>
#include <string>
#include <map>
#include <functional>
#include <stdexcept>

#include "initialize_od.cuh"
#include "kernels_od.cuh"
#include "kernels_fluidbcs.cuh"
#include "kernels_od_intvar.cuh"
#include "kernels_intvarbcs.cuh"

#include "utils.cuh"
#include "utils.hpp"

/* 
THIS CAN BE MOVED TO LIBRARIES 
*/
// I don't want to have a separate runtime file for each problem
class SimulationInitializer {
   private:
       using KernelLauncher = std::function<void(float*, const InitConfig&)>;
       std::map<std::string, KernelLauncher> initFunctions;
       InitConfig config;
   
   public:
       SimulationInitializer(const InitConfig& config) : config(config) {
           initFunctions["screwpinch"] = [this](float* data, const InitConfig& cfg) {
               LaunchScrewPinch(data, cfg); // Do not want to pass cfg to GPU or make this code less readable by passing long list of cfg parameters
           };
           initFunctions["screwpinch-stride"] = [this](float* data, const InitConfig& cfg) {
               LaunchScrewPinchStride(data, cfg);
           };
           /* ADD OTHER INITIALIZERS */
       } 

       void initialize(const std::string& simType, float* data){
           auto it = initFunctions.find(simType);
           if (it == initFunctions.end()) {
               throw std::runtime_error("Unknown simulation type: " + simType);
           }
           it->second(data, config);
       }
};

/* 
THIS CAN BE MOVED TO LIBRARIES
*/
// I don't want to have a separate runtime file for each possible choice of megakernels / microkernels
// Due to structure, it looks like I will need to separate instances of this class
class KernelConfigurer {
   private:
      using KernelLauncher = std::function<void(float*, const float*, const KernelConfig& kcfg)>;
      std::map<std::string, KernelLauncher> kernelFunctions;
      KernelConfig config;

   public:
      KernelConfigurer(const KernelConfig& kcfg) : config(config) {
         kernelFunctions["fluidadvancelocal-nodiff"] = [this](float* fluidvars, const float *intvars, const KernelConfig& kcfg) {
            LaunchFluidAdvanceLocalNoDiff(fluidvars, intvars, kcfg); // Do not want to pass kcfg to GPU or make this code less readable by passing long list of params
         };
         /* ADD MORE BUNDLES OF KERNELS TO RUN */
      }

      void LaunchKernels(const std::string& kBundle, float* fvars_or_intvars, const float* intvars_or_fvars){
         auto it = kernelFunctions.find(kBundle);
         if (it == kernelFunctions.end()) {
            throw std::runtime_error("Unknown kernel bundle selected: " + kBundle);
         }
         it->second(fvars_or_intvars, intvars_or_fvars, config);
      }
};

int main(int argc, char* argv[]){
   std::string sim_type = argv[1];

   int Nt = atoi(argv[2]);
   int Nx = atoi(argv[3]);
   int Ny = atoi(argv[4]);
   int Nz = atoi(argv[5]);

   float J0 = atof(argv[6]); // amplitude of the current
   float D = atof(argv[7]); // diffusion
   float r_max_coeff = atof(argv[8]); // r_pinch = r_max_coeff * r_max
   
   float x_min = atof(argv[9]);
   float x_max = atof(argv[10]);
   float y_min = atof(argv[11]);
   float y_max = atof(argv[12]);
   float z_min = atof(argv[13]);
   float z_max = atof(argv[14]);
   float dt = atof(argv[15]);
   
   std::string path_to_data = argv[16];
   std::string phdf5_bin_name = argv[17];
   std::string attr_bin_name = argv[18];
   std::string write_grid_bin_name = argv[19];
   std::string eigen_bin_name = argv[20];
   std::string num_proc = argv[21];

   int xgrid_threads = atoi(argv[22]);
   int ygrid_threads = atoi(argv[23]);
   int zgrid_threads = atoi(argv[24]);

   int init_xthreads = atoi(argv[25]);
   int init_ythreads = atoi(argv[26]);
   int init_zthreads = atoi(argv[27]);
   
   int FA_xthreads = atoi(argv[28]);
   int FA_ythreads = atoi(argv[29]);
   int FA_zthreads = atoi(argv[30]);

   int BCLeftRight_xthreads = atoi(argv[31]);
   int BCLeftRight_zthreads = atoi(argv[32]);

   int BCTopBottom_ythreads = atoi(argv[33]);
   int BCTopBottom_zthreads = atoi(argv[34]);

   int PBC_xthreads = atoi(argv[35]);
   int PBC_ythreads = atoi(argv[36]);

   int QintBC_FrontRight_xthreads = atoi(argv[37]);
   int QintBC_FrontBottom_ythreads = atoi(argv[38]);
   int QintBC_BottomRight_zthreads = atoi(argv[39]);

   int SM_mult_grid_x = atoi(argv[40]);
   int SM_mult_grid_y = atoi(argv[41]);
   int SM_mult_grid_z = atoi(argv[42]);

   int SM_mult_init_x = atoi(argv[43]);
   int SM_mult_init_y = atoi(argv[44]);
   int SM_mult_init_z = atoi(argv[45]);

   int SM_mult_FA_x = atoi(argv[46]);
   int SM_mult_FA_y = atoi(argv[47]);
   int SM_mult_FA_z = atoi(argv[48]);

   // CUDA BOILERPLATE 
   int deviceId;
   int numberOfSMs;

   hipGetDevice(&deviceId); // number of blocks should be a multiple of the number of device SMs
   hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

   size_t cube_size = Nx * Ny * Nz;
   size_t fluidvar_size = sizeof(float) * cube_size;
   size_t fluid_data_size = 8 * fluidvar_size;

   float *fluidvars, *intvars;

   checkCuda(hipMalloc(&fluidvars, fluid_data_size));
   checkCuda(hipMalloc(&intvars, fluid_data_size));
   
   float *x_grid, *y_grid, *z_grid;

   checkCuda(hipMalloc(&x_grid, sizeof(float) * Nx));
   checkCuda(hipMalloc(&y_grid, sizeof(float) * Ny));
   checkCuda(hipMalloc(&z_grid, sizeof(float) * Nz));

   float dx = (x_max - x_min) / (Nx - 1);
	float dy = (y_max - y_min) / (Ny - 1);
	float dz = (z_max - z_min) / (Nz - 1);

   // Execution grid and threadblock configurations for the Grid Initialization megakernel
   // dim3 egd_grid(SM_mult_grid_x * numberOfSMs, SM_mult_grid_y * numberOfSMs, SM_mult_grid_z * numberOfSMs);
   // dim3 tbd_grid(xgrid_threads, ygrid_threads, zgrid_threads);

   // Execution grid and threadblock configurations for the Grid Initialization microkernels
   dim3 egd_xgrid(SM_mult_grid_x * numberOfSMs, 1, 1); // "egd" = "execution_grid_dimensions"
   dim3 egd_ygrid(1, SM_mult_grid_y * numberOfSMs, 1);
   dim3 egd_zgrid(1, 1, SM_mult_grid_z * numberOfSMs);

   dim3 tbd_xgrid(xgrid_threads, 1, 1); // "tbd = thread_block_dimensions"
   dim3 tbd_ygrid(1, ygrid_threads, 1);
   dim3 tbd_zgrid(1, 1, zgrid_threads);

   // Execution grid and threadblock configurations for the initialization kernels
   dim3 egd_init(SM_mult_init_x * numberOfSMs, SM_mult_init_y * numberOfSMs, SM_mult_init_z * numberOfSMs);
   dim3 tbd_init(init_xthreads, init_ythreads, init_zthreads); 

   // Execution grid and threadblock configurations for the Boundary Condition microkernels
   dim3 egd_bdry_leftright(numberOfSMs, 1, numberOfSMs);
   dim3 egd_bdry_topbottom(1, numberOfSMs, numberOfSMs);
   dim3 egd_bdry_frontback(numberOfSMs, numberOfSMs, 1);

   // Execution grid specification for the Qint BCs - they require specifying values along certain lines
   dim3 egd_qintbdry_frontright(numberOfSMs, 1, 1); 
   dim3 egd_qintbdry_frontbottom(1, numberOfSMs, 1);
   dim3 egd_qintbdry_bottomright(1, 1, numberOfSMs);
   
   dim3 tbd_bdry_leftright(8, 1, 8);
   dim3 tbd_bdry_topbottom(1, 8, 8);
   dim3 tbd_bdry_frontback(8, 8, 1); // can also be used for PBCs
   dim3 tbd_qintbdry_frontright(1024, 1, 1);
   dim3 tbd_qintbdry_frontbottom(1, 1024, 1);
   dim3 tbd_qintbdry_bottomright(1, 1, 1024);

   // Execution grid and threadblock configurations for the Predictor and Corrector kernels
   dim3 egd_fluidadvance(SM_mult_FA_x * numberOfSMs, SM_mult_FA_y * numberOfSMs, SM_mult_FA_z * numberOfSMs);
   dim3 tbd_fluidadvance(FA_xthreads, FA_ythreads, FA_zthreads); 
  
   // InitializeGrid<<<egd_grid, tbd_grid>>>(x_min, x_max, y_min, y_max, z_min, z_max, dx, dy, dz, x_grid, y_grid, z_grid, Nx, Ny, Nz);
   InitializeX<<<egd_xgrid, tbd_xgrid>>>(x_grid, x_min, dx, Nx);
   InitializeY<<<egd_ygrid, tbd_ygrid>>>(y_grid, y_min, dy, Ny);
   InitializeZ<<<egd_zgrid, tbd_zgrid>>>(z_grid, z_min, dz, Nz);
   // hipMemset(fluidvars, 0, sizeof(float) * Nx * Ny * Nz); /* Not sure if these are necessary */
   // hipMemset(intvars, 0, sizeof(float) * Nx * Ny * Nz);
   // ZeroVars<<<egd_init, tbd_init>>>(fluidvars, Nx, Ny, Nz); 
   // ZeroVars<<<egd_init, tbd_init>>>(intvars, Nx, Ny, Nz); 
   checkCuda(hipDeviceSynchronize());

   InitConfig initParameters;
   initParameters.gridDim = egd_init;
   initParameters.blockDim = tbd_init;

   initParameters.J0 = J0;
   initParameters.r_max_coeff = r_max_coeff;

   initParameters.x_grid = x_grid;
   initParameters.y_grid = y_grid;
   initParameters.z_grid = z_grid;

   initParameters.Nx = Nx;
   initParameters.Ny = Ny;
   initParameters.Nz = Nz;

   SimulationInitializer simInit(initParameters);

   simInit.initialize(sim_type, fluidvars);
   checkCuda(hipDeviceSynchronize());

   KernelConfig fluidKernelParameters; // For selecting different bundles of kernels to use, i.e., megakernel or ordered microkernels (for profiling) 

   fluidKernelParameters.gridDim = egd_fluidadvance;
   fluidKernelParameters.blockDim = tbd_fluidadvance;

   fluidKernelParameters.D = D;
   
   fluidKernelParameters.dt = dt;
   fluidKernelParameters.dx = dx;
   fluidKernelParameters.dy = dy;
   fluidKernelParameters.dz = dz;

   fluidKernelParameters.Nx = Nx;
   fluidKernelParameters.Ny = Ny;
   fluidKernelParameters.Nz = Nz;

   KernelConfigurer fluidKcfg(fluidKernelParameters);

   /* 
   THERE SHOULD BE A `class BCsConfigurer to test different ones! 
   */
   rigidConductingWallBCsLeftRight<<<egd_bdry_leftright, tbd_bdry_leftright>>>(fluidvars, Nx, Ny, Nz);
   rigidConductingWallBCsTopBottom<<<egd_bdry_topbottom, tbd_bdry_topbottom>>>(fluidvars, Nx, Ny, Nz);
   PBCs<<<egd_bdry_frontback, tbd_bdry_frontback>>>(fluidvars, Nx, Ny, Nz);
   checkCuda(hipDeviceSynchronize());

   /*
   NOTE: 
   If you want to use microkernels here, you have to come up with an execution configuration set, and addtl. synchronization 
   */
   /* REFACTOR TO HAVE A RUNTIME CLASS THAT DECIDES WHAT SET OF KERNELS TO USE */
   ComputeIntermediateVariablesNoDiff<<<egd_fluidadvance, tbd_fluidadvance>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
   checkCuda(hipDeviceSynchronize());    

   /*
   NOTE:
   You DEFINITELY want to use microkernels here
   */
   /* 
   REFACTOR TO HAVE A RUNTIME CLASS THAT DECIDES WHAT SET OF KERNELS TO USE 
   `class QintBCsConfigurer` 
   */
   QintBdryFrontNoDiff<<<egd_bdry_frontback, tbd_bdry_frontback>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
   QintBdryLeftRightNoDiff<<<egd_bdry_leftright, tbd_bdry_leftright>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
   QintBdryTopBottomNoDiff<<<egd_bdry_topbottom, tbd_bdry_topbottom>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
   QintBdryFrontBottomNoDiff<<<egd_qintbdry_frontbottom, tbd_qintbdry_frontbottom>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
   QintBdryFrontRightNoDiff<<<egd_qintbdry_frontright, tbd_qintbdry_frontright>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
   QintBdryBottomRightNoDiff<<<egd_qintbdry_bottomright, tbd_qintbdry_bottomright>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
   checkCuda(hipDeviceSynchronize());    

   QintBdryPBCs<<<egd_bdry_frontback, tbd_bdry_frontback>>>(fluidvars, intvars, Nx, Ny, Nz);
   checkCuda(hipDeviceSynchronize());    

   /* 
   REFACTOR TO HAVE A WRAPPER 
   Argument is `std::string shm_name_var`
   */
   // Use IPC to write data out in order to avoid redundant work 
   std::string shm_name_fluidvar = "/shared_h_fluidvar";
   int shm_fd = shm_open(shm_name_fluidvar.data(), O_CREAT | O_RDWR, 0666);
   if (shm_fd == -1) {
      std::cerr << "Failed to create shared memory!" << std::endl;
      return EXIT_FAILURE;
   }
   
   ftruncate(shm_fd, fluid_data_size);
   float* shm_h_fluidvar = (float*)mmap(0, fluid_data_size, PROT_READ | PROT_WRITE, MAP_SHARED, shm_fd, 0);
   if (shm_h_fluidvar == MAP_FAILED) {
      std::cerr << "mmap failed!" << std::endl;
      return EXIT_FAILURE;
   }

   std::cout << "Transferring device data to host" << std::endl;
   hipMemcpy(shm_h_fluidvar, fluidvars, fluid_data_size, hipMemcpyDeviceToHost);
   checkCuda(hipDeviceSynchronize());

   std::string filename_fluidvar = path_to_data + "fluidvars_0.h5";

   std::cout << "Writing Screw-Pinch ICs out with PHDF5" << std::endl;
   int ret = callBinary_PHDF5Write(filename_fluidvar, Nx, Ny, Nz, shm_name_fluidvar, fluid_data_size, num_proc, phdf5_bin_name); 
   if (ret != 0) {
        std::cerr << "Error executing PHDF5 command" << std::endl;
   }

   std::cout << "Writing attributes to the dataset with HDF5" << std::endl;
   ret = callBinary_AttrWrite(filename_fluidvar, Nx, Ny, Nz, attr_bin_name); // inadvisable to write attributes in a PHDF5 context
   if (ret != 0) {
        std::cerr << "Error executing attribute command" << std::endl;
   }

   // COMPUTE STABILITY CRITERION
   /* REFACTOR TO BASE ON ANALYTIC EXPRESSIONS FOR EIGNEVALUES */
   // First, transfer grid data
   std::string shm_name_gridx = "/shared_h_gridx";
   shm_fd = shm_open(shm_name_gridx.data(), O_CREAT | O_RDWR, 0666);
   ftruncate(shm_fd, sizeof(float) * Nx);
   float* shm_h_gridx = (float*)mmap(0, sizeof(float) * Nx, PROT_READ | PROT_WRITE, MAP_SHARED, shm_fd, 0);
   if (shm_h_gridx == MAP_FAILED) {
      std::cerr << "mmap failed for grid_x!" << std::endl;
      return EXIT_FAILURE;
   }
   hipMemcpy(shm_h_gridx, x_grid, sizeof(float) * Nx, hipMemcpyDeviceToHost);

   std::string shm_name_gridy = "/shared_h_gridy";
   shm_fd = shm_open(shm_name_gridy.data(), O_CREAT | O_RDWR, 0666);
   ftruncate(shm_fd, sizeof(float) * Ny);
   float* shm_h_gridy = (float*)mmap(0, sizeof(float) * Ny, PROT_READ | PROT_WRITE, MAP_SHARED, shm_fd, 0);
   if (shm_h_gridy == MAP_FAILED) {
      std::cerr << "mmap failed for grid_y!" << std::endl;
      return EXIT_FAILURE;
   }
   hipMemcpy(shm_h_gridy, y_grid, sizeof(float) * Ny, hipMemcpyDeviceToHost);

   std::string shm_name_gridz = "/shared_h_gridz";
   shm_fd = shm_open(shm_name_gridz.data(), O_CREAT | O_RDWR, 0666);
   ftruncate(shm_fd, sizeof(float) * Nz);
   float* shm_h_gridz = (float*)mmap(0, sizeof(float) * Nz, PROT_READ | PROT_WRITE, MAP_SHARED, shm_fd, 0);
   if (shm_h_gridz == MAP_FAILED) {
      std::cerr << "mmap failed for grid_z!" << std::endl;
      return EXIT_FAILURE;
   }
   hipMemcpy(shm_h_gridz, z_grid, sizeof(float) * Nz, hipMemcpyDeviceToHost);
   checkCuda(hipDeviceSynchronize());

   std::cout << "Forking to process for writing grid to storage" << std::endl;
   ret = callBinary_WriteGrid(write_grid_bin_name, path_to_data, shm_name_gridx, shm_name_gridy, shm_name_gridz, Nx, Ny, Nz);
   if (ret != 0) {
         std::cerr << "Error executing writegrid binary: " << eigen_bin_name << std::endl;
   }

   if (!(eigen_bin_name == "none")){ // Don't always want to check stability - expensive raster scan
      std::cout << "Forking to process for computing CFL number (checking stability)" << std::endl;
      ret = callBinary_EigenSC(shm_name_fluidvar, Nx, Ny, Nz, eigen_bin_name, dt, dx, dy, dz, shm_name_gridx, shm_name_gridy, shm_name_gridz);
      if (ret != 0) {
         std::cerr << "Error executing Eigen binary: " << eigen_bin_name << std::endl;
         std::cerr << "Error code: " << ret << std::endl;
      }
   }

   // SIMULATION LOOP
   for (int it = 1; it < Nt; it++){
      std::cout << "Starting timestep " << it << std::endl;

      std::cout << "Launching megakernel for computing fluid variables" << std::endl;
      FluidAdvanceLocalNoDiff<<<egd_fluidadvance, tbd_fluidadvance>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
      checkCuda(hipDeviceSynchronize());
      
      std::cout << "Launching microkernel for PBCs" << std::endl; 
      PBCs<<<egd_bdry_frontback, tbd_bdry_frontback>>>(fluidvars, Nx, Ny, Nz);
      checkCuda(hipDeviceSynchronize());
      std::cout << "Kernels for computing fluid variables completed" << std::endl;
      
      std::cout << "Launching megakernel for computing intermediate variables" << std::endl; 
      /* NOTE: Thrashes the cache */
      ComputeIntermediateVariablesNoDiff<<<egd_fluidadvance, tbd_fluidadvance>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
      checkCuda(hipDeviceSynchronize());

      std::cout << "Launching microkernels for computing Qint boundaries" << std::endl; 
      QintBdryFrontNoDiff<<<egd_bdry_frontback, tbd_bdry_frontback>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
      QintBdryLeftRightNoDiff<<<egd_bdry_leftright, tbd_bdry_leftright>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
      QintBdryTopBottomNoDiff<<<egd_bdry_topbottom, tbd_bdry_topbottom>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
      QintBdryFrontBottomNoDiff<<<egd_qintbdry_frontbottom, tbd_qintbdry_frontbottom>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
      QintBdryFrontRightNoDiff<<<egd_qintbdry_frontright, tbd_qintbdry_frontright>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
      QintBdryBottomRightNoDiff<<<egd_qintbdry_bottomright, tbd_qintbdry_bottomright>>>(fluidvars, intvars, dt, dx, dy, dz, Nx, Ny, Nz);
      checkCuda(hipDeviceSynchronize());    

      std::cout << "Launching kernel for computing Qint PBCs" << std::endl; 
      QintBdryPBCs<<<egd_bdry_frontback, tbd_bdry_frontback>>>(fluidvars, intvars, Nx, Ny, Nz);
      checkCuda(hipDeviceSynchronize());    

      std::cout << "Transferring updated fluid data to host" << std::endl;
      hipMemcpy(shm_h_fluidvar, fluidvars, fluid_data_size, hipMemcpyDeviceToHost);
      checkCuda(hipDeviceSynchronize());
      std::cout << "Kernels for computing intermediate variables completed" << std::endl;
      std::cout << "Fluid D2H data migration completed" << std::endl;

      std::cout << "Writing updated fluid data out" << std::endl;
      filename_fluidvar = path_to_data + "fluidvars_" + std::to_string(it) + ".h5";
      ret = callBinary_PHDF5Write(filename_fluidvar, Nx, Ny, Nz, shm_name_fluidvar, fluid_data_size, num_proc, phdf5_bin_name);
      if (ret != 0) {
         std::cerr << "Error forking PHDF5Write binary on timestep " << std::to_string(it) << std::endl;
      }  

      std::cout << "Timestep " << it << " complete" << std::endl;

      if (!(eigen_bin_name == "none")){ // Don't always want to check stability - expensive raster scan
         std::cout << "Forking to process for computing CFL number (checking stability)" << std::endl;
         ret = callBinary_EigenSC(shm_name_fluidvar, Nx, Ny, Nz, eigen_bin_name, dt, dx, dy, dz, shm_name_gridx, shm_name_gridy, shm_name_gridz);
         if (ret != 0) {
            std::cerr << "Error executing Eigen binary: " << eigen_bin_name << std::endl;
            std::cerr << "Error code: " << ret << std::endl;
         }
      }  
   } 

   // FREE EVERYTHING
   // Device
   checkCuda(hipFree(fluidvars));
   checkCuda(hipFree(intvars));
   checkCuda(hipFree(x_grid));
   checkCuda(hipFree(y_grid));
   checkCuda(hipFree(z_grid));
   // checkCuda(hipFree(d_initParameters));
   
   // Host
   munmap(shm_h_fluidvar, 8 * fluid_data_size);
   munmap(shm_h_gridx, sizeof(float) * Nx);
   munmap(shm_h_gridy, sizeof(float) * Ny);
   munmap(shm_h_gridz, sizeof(float) * Nz);
   shm_unlink(shm_name_fluidvar.data());
   shm_unlink(shm_name_gridx.data());
   shm_unlink(shm_name_gridy.data());
   shm_unlink(shm_name_gridz.data());
   return 0;
}