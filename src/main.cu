#include <assert.h>
#include <cstdlib>
#include <stdio.h>
#include <iostream>

#include "../include/kernels_od.cuh"
#include "../include/initialize_od.cuh"

// #include "../include/kernel_od.cu"

/* KERNEL FOR TRANSFERRING DATA FROM FUTURE TIMESTEP TO CURRENT */

// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define checkCuda(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char* argv[]){
	// Parse inputs - provided by imhdLauncher.py because Python parsing is easiest, and Python launching is easy
	int Nt = atoi(argv[1]);
	int Nx = atoi(argv[2]);
	int Ny = atoi(argv[3]);
	int Nz = atoi(argv[4]);
	int SM_mult_x = atoi(argv[5]);
	int SM_mult_y = atoi(argv[6]);
	int SM_mult_z = atoi(argv[7]);
	int num_threads_per_block_x = atoi(argv[8]);
	int num_threads_per_block_y = atoi(argv[9]);
	int num_threads_per_block_z = atoi(argv[10]);
	float J0 = atof(argv[11]);
	float D = atof(argv[12]);
	float x_min = atof(argv[13]);
	float x_max = atof(argv[14]);
	float y_min = atof(argv[15]);
	float y_max = atof(argv[16]);
	float z_min = atof(argv[17]);
	float z_max = atof(argv[18]);
	float dt = atof(argv[19]);

	float dx = (x_max - x_min) / (Nx - 1);
	float dy = (y_max - y_min) / (Ny - 1);
	float dz = (z_max - z_min) / (Nz - 1);

	/* Initialize device data */
	int deviceId;
	int numberOfSMs;

	hipGetDevice(&deviceId);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

	float *rho, *rhov_x, *rhov_y, *rhov_z, *Bx, *By, *Bz, *e;
	float *rho_np1, *rhovx_np1, *rhovy_np1, *rhovz_np1, *Bx_np1, *By_np1, *Bz_np1, *e_np1;
	float *rho_int, *rhovx_int, *rhovy_int, *rhovz_int, *Bx_int, *By_int, *Bz_int, *e_int;
	float *grid_x, *grid_y, *grid_z;

	dim3 grid_dimensions(SM_mult_x * numberOfSMs, SM_mult_y * numberOfSMs, SM_mult_z * numberOfSMs);
	dim3 block_dimensions(num_threads_per_block_x, num_threads_per_block_y, num_threads_per_block_z);

	InitializeGrid<<<grid_dimensions, block_dimensions>>>(x_min, x_max, y_min, y_max, z_min, z_max, 
															grid_x, grid_y, grid_z, Nx, Ny, Nz);
	checkCuda(hipDeviceSynchronize());

	InitialConditions<<<grid_dimensions, block_dimensions>>>(rho, rhov_x, rhov_y, rhov_z, Bx, By, Bz, e, 
																J0, grid_x, grid_y, grid_z, Nx, Ny, Nz);
	checkCuda(hipDeviceSynchronize());

	/* Simulation loop */
	for (int it = 0; it < Nt; it++){
		/* Write data out - use GPUDirect Storage (GDS) */

		/* Compute interior and boundaries*/
		FluidAdvance<<<grid_dimensions, block_dimensions>>>(rho_np1, rhovx_np1, rhovy_np1, rhovz_np1, Bx_np1, By_np1, Bz_np1, e_np1, 
																rho, rhov_x, rhov_y, rhov_z, Bx, By, Bz, e, 
																rho_int, rhovx_int, rhovy_int, rhovz_int, Bx_int, By_int, Bz_int, e_int, 
																D, dt, dx, dy, dz, Nx, Ny, Nz);
		BoundaryConditions<<<grid_dimensions, block_dimensions>>>(rho_np1, rhovx_np1, rhovy_np1, rhovz_np1, Bx_np1, By_np1, Bz_np1, e_np1,
																	rho, rhov_x, rhov_y, rhov_z, Bx, By, Bz, e, Nx, Ny, Nz);
		checkCuda(hipDeviceSynchronize());
		/* Transfer future timestep data to current timestep */
		checkCuda(hipDeviceSynchronize());
	}

	/* Free */ 
	return 0;
}