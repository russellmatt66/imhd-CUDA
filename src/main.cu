#include <assert.h>
#include <cstdlib>
#include <stdio.h>
#include <iostream>

#include "../include/kernels_od.cuh"
#include "../include/initialize_od.cuh"
#include "../include/gds.cuh"
#include "../include/utils.hpp"

// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define checkCuda(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char* argv[]){
	// Parse inputs - provided by imhdLauncher.py because Python parsing is easiest, and Python launching is easy
	int Nt = atoi(argv[1]);
	int Nx = atoi(argv[2]);
	int Ny = atoi(argv[3]);
	int Nz = atoi(argv[4]);
	int SM_mult_x = atoi(argv[5]);
	int SM_mult_y = atoi(argv[6]);
	int SM_mult_z = atoi(argv[7]);
	int num_threads_per_block_x = atoi(argv[8]);
	int num_threads_per_block_y = atoi(argv[9]);
	int num_threads_per_block_z = atoi(argv[10]);
	float J0 = atof(argv[11]);
	float D = atof(argv[12]);
	float x_min = atof(argv[13]);
	float x_max = atof(argv[14]);
	float y_min = atof(argv[15]);
	float y_max = atof(argv[16]);
	float z_min = atof(argv[17]);
	float z_max = atof(argv[18]);
	float dt = atof(argv[19]);
	int write_rho = atoi(argv[20]); // Data volume gets very large
	int write_rhovx = atoi(argv[21]);
	int write_rhovy = atoi(argv[22]);
	int write_rhovz = atoi(argv[23]);
	int write_Bx = atoi(argv[24]);
	int write_By = atoi(argv[25]);
	int write_Bz = atoi(argv[26]);
	int write_e = atoi(argv[27]);

	float dx = (x_max - x_min) / (Nx - 1);
	float dy = (y_max - y_min) / (Ny - 1);
	float dz = (z_max - z_min) / (Nz - 1);

	// Initialize device data
	int deviceId;
	int numberOfSMs;

	hipGetDevice(&deviceId);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

	int* to_write_or_not;
	to_write_or_not = (int*)malloc(8 * sizeof(int));

	for (int i = 0; i < 8; i++){ /* COULD USE A CHAR FOR THIS */
		to_write_or_not[i] = atoi(argv[20 + i]);
	}

	float *rho, *rhov_x, *rhov_y, *rhov_z, *Bx, *By, *Bz, *e;
	float *rho_np1, *rhovx_np1, *rhovy_np1, *rhovz_np1, *Bx_np1, *By_np1, *Bz_np1, *e_np1;
	float *rho_int, *rhovx_int, *rhovy_int, *rhovz_int, *Bx_int, *By_int, *Bz_int, *e_int;
	float *grid_x, *grid_y, *grid_z;

	int fluid_data_size = sizeof(float) * Nx * Ny * Nz;

	/* MALLOC TO DEVICE */
	checkCuda(hipMalloc(&rho, fluid_data_size));
	checkCuda(hipMalloc(&rhov_x, fluid_data_size));
	checkCuda(hipMalloc(&rhov_y, fluid_data_size));
	checkCuda(hipMalloc(&rhov_z, fluid_data_size));
	checkCuda(hipMalloc(&Bx, fluid_data_size));
	checkCuda(hipMalloc(&By, fluid_data_size));
	checkCuda(hipMalloc(&Bz, fluid_data_size));
	checkCuda(hipMalloc(&e, fluid_data_size));

	checkCuda(hipMalloc(&rho_np1, fluid_data_size));
	checkCuda(hipMalloc(&rhovx_np1, fluid_data_size));
	checkCuda(hipMalloc(&rhovy_np1, fluid_data_size));
	checkCuda(hipMalloc(&rhovz_np1, fluid_data_size));
	checkCuda(hipMalloc(&Bx_np1, fluid_data_size));
	checkCuda(hipMalloc(&By_np1, fluid_data_size));
	checkCuda(hipMalloc(&Bz_np1, fluid_data_size));
	checkCuda(hipMalloc(&e_np1, fluid_data_size));

	checkCuda(hipMalloc(&rho_int, fluid_data_size));
	checkCuda(hipMalloc(&rhovx_int, fluid_data_size));
	checkCuda(hipMalloc(&rhovy_int, fluid_data_size));
	checkCuda(hipMalloc(&rhovz_int, fluid_data_size));
	checkCuda(hipMalloc(&Bx_int, fluid_data_size));
	checkCuda(hipMalloc(&By_int, fluid_data_size));
	checkCuda(hipMalloc(&Bz_int, fluid_data_size));
	checkCuda(hipMalloc(&e_int, fluid_data_size));

	checkCuda(hipMalloc(&grid_x, sizeof(float) * Nx));
	checkCuda(hipMalloc(&grid_y, sizeof(float) * Ny));
	checkCuda(hipMalloc(&grid_z, sizeof(float) * Nz));

	dim3 grid_dimensions(SM_mult_x * numberOfSMs, SM_mult_y * numberOfSMs, SM_mult_z * numberOfSMs);
	dim3 block_dimensions(num_threads_per_block_x, num_threads_per_block_y, num_threads_per_block_z);

	InitializeGrid<<<grid_dimensions, block_dimensions>>>(x_min, x_max, y_min, y_max, z_min, z_max, dx, dy, dz,
															grid_x, grid_y, grid_z, Nx, Ny, Nz);
	checkCuda(hipDeviceSynchronize());

	InitialConditions<<<grid_dimensions, block_dimensions>>>(rho, rhov_x, rhov_y, rhov_z, Bx, By, Bz, e, 
																J0, grid_x, grid_y, grid_z, Nx, Ny, Nz); // Screw-pinch
	InitializeIntAndSwap<<<grid_dimensions, block_dimensions>>>(rho_np1, rhovx_np1, rhovy_np1, rhovz_np1, Bx_np1, By_np1, Bz_np1, e_np1,
																rho_int, rhovx_int, rhovy_int, rhovz_int, Bx_int, By_int, Bz_int, e_int, 
																Nx, Ny, Nz); // All 0.0
	checkCuda(hipDeviceSynchronize());

    // Prepare host data for writing out
	std::vector<std::string> fluid_data_files (8); // 8 is the number of threads I'm going with
    std::string base_file = "../data/rho/";
    for (size_t i = 0; i < fluid_data_files.size(); i++){
        fluid_data_files[i] = base_file + std::to_string(i) + ".csv";
    }   

	float *h_rho, *h_rhovx, *h_rhovy, *h_rhovz, *h_Bx, *h_By, *h_Bz, *h_e;

	h_rho = (float*)malloc(fluid_data_size);
	h_rhovx = (float*)malloc(fluid_data_size);
	h_rhovy = (float*)malloc(fluid_data_size);
	h_rhovz = (float*)malloc(fluid_data_size);
	h_Bx = (float*)malloc(fluid_data_size);
	h_By = (float*)malloc(fluid_data_size);
	h_Bz = (float*)malloc(fluid_data_size);
	h_e = (float*)malloc(fluid_data_size);

	for (size_t ih = 0; ih < 8; ih++){
		if (!to_write_or_not[ih]){ // No need for the host memory if it's not being written out
			switch (ih)
			{
			case 0:
				free(h_rho);
				break;
			case 1:
				free(h_rhovx);
				break;
			case 2:
				free(h_rhovy);
				break;			
			case 3:
				free(h_rhovz);
				break;			
			case 4:
				free(h_Bx);
				break;			
			case 5:
				free(h_By);
				break;			
			case 6:
				free(h_Bz);
				break;			
			case 7:
				free(h_e);
				break;			
			default:
				break;
			}
		}
	}

	/* Simulation loop */
	for (size_t it = 0; it < Nt; it++){
		std::cout << "Starting iteration " << it << std::endl;

		/* Compute interior and boundaries*/
		std::cout << "Evolving fluid interior and boundary" << std::endl; 
		FluidAdvance<<<grid_dimensions, block_dimensions>>>(rho_np1, rhovx_np1, rhovy_np1, rhovz_np1, Bx_np1, By_np1, Bz_np1, e_np1, 
																rho, rhov_x, rhov_y, rhov_z, Bx, By, Bz, e, 
																rho_int, rhovx_int, rhovy_int, rhovz_int, Bx_int, By_int, Bz_int, e_int, 
																D, dt, dx, dy, dz, Nx, Ny, Nz);
		BoundaryConditions<<<grid_dimensions, block_dimensions>>>(rho_np1, rhovx_np1, rhovy_np1, rhovz_np1, Bx_np1, By_np1, Bz_np1, e_np1,
																	rho, rhov_x, rhov_y, rhov_z, Bx, By, Bz, e, 
																	rho_int, rhovx_int, rhovy_int, rhovz_int, Bx_int, By_int, Bz_int, e_int, 
																	D, dt, dx, dy, dz, Nx, Ny, Nz);
	
		std::cout << "Writing fluid data to host" << std::endl;
		// Data volume scales very fast w/problem size, don't want to always write everything out 
		for (size_t iv = 0; iv < 8; iv++){ 
			if (to_write_or_not[iv]){  
				switch (iv)
				{
				case 0:
					hipMemcpy(h_rho, rho, fluid_data_size, hipMemcpyDeviceToHost);
					break;
				case 1:
					hipMemcpy(h_rhovx, rhov_x, fluid_data_size, hipMemcpyDeviceToHost);
					break;
				case 2:
					hipMemcpy(h_rhovy, rhov_y, fluid_data_size, hipMemcpyDeviceToHost);
					break;
				case 3:
					hipMemcpy(h_rhovz, rhov_z, fluid_data_size, hipMemcpyDeviceToHost);
					break;			
				case 4:
					hipMemcpy(h_Bx, Bx, fluid_data_size, hipMemcpyDeviceToHost);
					break;				
				case 5:
					hipMemcpy(h_By, By, fluid_data_size, hipMemcpyDeviceToHost);
					break;				
				case 6:
					hipMemcpy(h_Bz, Bz, fluid_data_size, hipMemcpyDeviceToHost);
					break;				
				case 7:
					hipMemcpy(h_e, e, fluid_data_size, hipMemcpyDeviceToHost);
					break;				
				default:
					break;
				}
			}
		}
		checkCuda(hipDeviceSynchronize());
		
		// Transfer future timestep data to current timestep in order to avoid race conditions
		std::cout << "Swapping future timestep to current" << std::endl;
		SwapSimData<<<grid_dimensions, block_dimensions>>>(rho, rhov_x, rhov_y, rhov_z, Bx, By, Bz, e, 
															rho_np1, rhovx_np1, rhovy_np1, rhovz_np1, Bx_np1, By_np1, Bz_np1, e_np1,
															Nx, Ny, Nz);

		// Split the Device2Host and Host2Storage writes up to reduce synchro barriers
		std::cout << "Writing host data to storage" << std::endl; 
		for (size_t iv = 0; iv < 8; iv++){ 
			if (to_write_or_not[iv]){ 
				base_file = getNewBaseDataLoc(iv);
				for (size_t i = 0; i < fluid_data_files.size(); i++){
					fluid_data_files[i] = base_file + std::to_string(i) + ".csv";
				}  
				switch (iv)
				{
				case 0:
					writeFluidVars(fluid_data_files, h_rho, Nx, Ny, Nz);					
					break;
				case 1:
					writeFluidVars(fluid_data_files, h_rhovx, Nx, Ny, Nz);					
					break;
				case 2:
					writeFluidVars(fluid_data_files, h_rhovy, Nx, Ny, Nz);					
					break;
				case 3:
					writeFluidVars(fluid_data_files, h_rhovz, Nx, Ny, Nz);					
					break;			
				case 4:
					writeFluidVars(fluid_data_files, h_Bx, Nx, Ny, Nz);					
					break;				
				case 5:
					writeFluidVars(fluid_data_files, h_By, Nx, Ny, Nz);					
					break;				
				case 6:
					writeFluidVars(fluid_data_files, h_Bz, Nx, Ny, Nz);					
					break;				
				case 7:
					writeFluidVars(fluid_data_files, h_e, Nx, Ny, Nz);					
					break;				
				default:
					break;
				}
			}
		}
		checkCuda(hipDeviceSynchronize());
	}

	/* Free device data */ 
	checkCuda(hipFree(rho));
	checkCuda(hipFree(rhov_x));
	checkCuda(hipFree(rhov_y));
	checkCuda(hipFree(rhov_z));
	checkCuda(hipFree(Bx));
	checkCuda(hipFree(By));
	checkCuda(hipFree(Bz));
	checkCuda(hipFree(e));

	checkCuda(hipFree(rho_np1));
	checkCuda(hipFree(rhovx_np1));
	checkCuda(hipFree(rhovy_np1));
	checkCuda(hipFree(rhovz_np1));
	checkCuda(hipFree(Bx_np1));
	checkCuda(hipFree(By_np1));
	checkCuda(hipFree(Bz_np1));
	checkCuda(hipFree(e_np1));

	checkCuda(hipFree(rho_int));
	checkCuda(hipFree(rhovx_int));
	checkCuda(hipFree(rhovy_int));
	checkCuda(hipFree(rhovz_int));
	checkCuda(hipFree(Bx_int));
	checkCuda(hipFree(By_int));
	checkCuda(hipFree(Bz_int));
	checkCuda(hipFree(e_int));

	checkCuda(hipFree(grid_x));
	checkCuda(hipFree(grid_y));
	checkCuda(hipFree(grid_z));

	/* Free host data */
	for (size_t ih = 0; ih < 8; ih++){
		if (to_write_or_not[ih]){ // Don't forget to free the rest of the host buffers 
			switch (ih)
			{
			case 0:
				free(h_rho);
				break;
			case 1:
				free(h_rhovx);
				break;
			case 2:
				free(h_rhovy);
				break;			
			case 3:
				free(h_rhovz);
				break;			
			case 4:
				free(h_Bx);
				break;			
			case 5:
				free(h_By);
				break;			
			case 6:
				free(h_Bz);
				break;			
			case 7:
				free(h_e);
				break;			
			default:
				break;
			}
		}
	}
	free(to_write_or_not);
	return 0;
}