#include <assert.h>
#include <cstdlib>
#include <stdio.h>
#include <iostream>

#include "../../include/kernels_od.cuh"
#include "../../include/initialize_od.cuh"
#include "../../include/gds.cuh"
#include "../../include/utils.hpp"

// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define checkCuda(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char* argv[]){
	// Parse inputs - provided by imhdLauncher.py because Python parsing is easiest, and Python launching is easy
    std::vector<float> inputs (19, 0.0);
    parseInputFileDebug(inputs, "./debug.inp");
	
    int Nt = int(inputs[0]);
	int Nx = int(inputs[1]);
	int Ny = int(inputs[2]);
	int Nz = int(inputs[3]);
	int SM_mult_x = int(inputs[4]);
	int SM_mult_y = int(inputs[5]);
	int SM_mult_z = int(inputs[6]);
	int num_threads_per_block_x = int(inputs[7]);
	int num_threads_per_block_y = int(inputs[8]);
	int num_threads_per_block_z = int(inputs[9]);
	float J0 = inputs[10];
	float D = inputs[11];
	float x_min = inputs[12];
	float x_max = inputs[13];
	float y_min = inputs[14];
	float y_max = inputs[15];
	float z_min = inputs[16];
	float z_max = inputs[17];
	float dt = inputs[18];

	float dx = (x_max - x_min) / (Nx - 1);
	float dy = (y_max - y_min) / (Ny - 1);
	float dz = (z_max - z_min) / (Nz - 1);

	// Initialize device data
	int deviceId;
	int numberOfSMs;

	hipGetDevice(&deviceId);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

	float *rho, *rhov_x, *rhov_y, *rhov_z, *Bx, *By, *Bz, *e;
	float *rho_np1, *rhovx_np1, *rhovy_np1, *rhovz_np1, *Bx_np1, *By_np1, *Bz_np1, *e_np1;
	float *rho_int, *rhovx_int, *rhovy_int, *rhovz_int, *Bx_int, *By_int, *Bz_int, *e_int;
	float *grid_x, *grid_y, *grid_z;

	int fluid_data_size = sizeof(float) * Nx * Ny * Nz;

	/* MALLOC TO DEVICE */
	checkCuda(hipMalloc(&rho, fluid_data_size));
	checkCuda(hipMalloc(&rhov_x, fluid_data_size));
	checkCuda(hipMalloc(&rhov_y, fluid_data_size));
	checkCuda(hipMalloc(&rhov_z, fluid_data_size));
	checkCuda(hipMalloc(&Bx, fluid_data_size));
	checkCuda(hipMalloc(&By, fluid_data_size));
	checkCuda(hipMalloc(&Bz, fluid_data_size));
	checkCuda(hipMalloc(&e, fluid_data_size));

	checkCuda(hipMalloc(&rho_np1, fluid_data_size));
	checkCuda(hipMalloc(&rhovx_np1, fluid_data_size));
	checkCuda(hipMalloc(&rhovy_np1, fluid_data_size));
	checkCuda(hipMalloc(&rhovz_np1, fluid_data_size));
	checkCuda(hipMalloc(&Bx_np1, fluid_data_size));
	checkCuda(hipMalloc(&By_np1, fluid_data_size));
	checkCuda(hipMalloc(&Bz_np1, fluid_data_size));
	checkCuda(hipMalloc(&e_np1, fluid_data_size));

	checkCuda(hipMalloc(&rho_int, fluid_data_size));
	checkCuda(hipMalloc(&rhovx_int, fluid_data_size));
	checkCuda(hipMalloc(&rhovy_int, fluid_data_size));
	checkCuda(hipMalloc(&rhovz_int, fluid_data_size));
	checkCuda(hipMalloc(&Bx_int, fluid_data_size));
	checkCuda(hipMalloc(&By_int, fluid_data_size));
	checkCuda(hipMalloc(&Bz_int, fluid_data_size));
	checkCuda(hipMalloc(&e_int, fluid_data_size));

	checkCuda(hipMalloc(&grid_x, sizeof(float) * Nx));
	checkCuda(hipMalloc(&grid_y, sizeof(float) * Ny));
	checkCuda(hipMalloc(&grid_z, sizeof(float) * Nz));

	dim3 grid_dimensions(SM_mult_x * numberOfSMs, SM_mult_y * numberOfSMs, SM_mult_z * numberOfSMs);
	dim3 block_dimensions(num_threads_per_block_x, num_threads_per_block_y, num_threads_per_block_z);

	InitializeGrid<<<grid_dimensions, block_dimensions>>>(x_min, x_max, y_min, y_max, z_min, z_max, dx, dy, dz,
															grid_x, grid_y, grid_z, Nx, Ny, Nz);
	checkCuda(hipDeviceSynchronize());

	InitialConditions<<<grid_dimensions, block_dimensions>>>(rho, rhov_x, rhov_y, rhov_z, Bx, By, Bz, e, 
																J0, grid_x, grid_y, grid_z, Nx, Ny, Nz); // Screw-pinch
	InitializeIntAndSwap<<<grid_dimensions, block_dimensions>>>(rho_np1, rhovx_np1, rhovy_np1, rhovz_np1, Bx_np1, By_np1, Bz_np1, e_np1,
																rho_int, rhovx_int, rhovy_int, rhovz_int, Bx_int, By_int, Bz_int, e_int, 
																Nx, Ny, Nz); // All 0.0
	checkCuda(hipDeviceSynchronize());

	/* Simulation loop */
	for (size_t it = 0; it < Nt; it++){
		std::cout << "Starting iteration " << it << std::endl;

		/* Compute interior and boundaries*/
		std::cout << "Evolving fluid interior and boundary" << std::endl; 
		FluidAdvance<<<grid_dimensions, block_dimensions>>>(rho_np1, rhovx_np1, rhovy_np1, rhovz_np1, Bx_np1, By_np1, Bz_np1, e_np1, 
																rho, rhov_x, rhov_y, rhov_z, Bx, By, Bz, e, 
																rho_int, rhovx_int, rhovy_int, rhovz_int, Bx_int, By_int, Bz_int, e_int, 
																D, dt, dx, dy, dz, Nx, Ny, Nz);
		BoundaryConditions<<<grid_dimensions, block_dimensions>>>(rho_np1, rhovx_np1, rhovy_np1, rhovz_np1, Bx_np1, By_np1, Bz_np1, e_np1,
																	rho, rhov_x, rhov_y, rhov_z, Bx, By, Bz, e, 
																	rho_int, rhovx_int, rhovy_int, rhovz_int, Bx_int, By_int, Bz_int, e_int, 
																	D, dt, dx, dy, dz, Nx, Ny, Nz);
	
		std::cout << "Writing fluid data to host" << std::endl;
		checkCuda(hipDeviceSynchronize());
		
		// Transfer future timestep data to current timestep in order to avoid race conditions
		std::cout << "Swapping future timestep to current" << std::endl;
		SwapSimData<<<grid_dimensions, block_dimensions>>>(rho, rhov_x, rhov_y, rhov_z, Bx, By, Bz, e, 
															rho_np1, rhovx_np1, rhovy_np1, rhovz_np1, Bx_np1, By_np1, Bz_np1, e_np1,
															Nx, Ny, Nz);
		checkCuda(hipDeviceSynchronize());
	}

	/* Free device data */ 
	checkCuda(hipFree(rho));
	checkCuda(hipFree(rhov_x));
	checkCuda(hipFree(rhov_y));
	checkCuda(hipFree(rhov_z));
	checkCuda(hipFree(Bx));
	checkCuda(hipFree(By));
	checkCuda(hipFree(Bz));
	checkCuda(hipFree(e));

	checkCuda(hipFree(rho_np1));
	checkCuda(hipFree(rhovx_np1));
	checkCuda(hipFree(rhovy_np1));
	checkCuda(hipFree(rhovz_np1));
	checkCuda(hipFree(Bx_np1));
	checkCuda(hipFree(By_np1));
	checkCuda(hipFree(Bz_np1));
	checkCuda(hipFree(e_np1));

	checkCuda(hipFree(rho_int));
	checkCuda(hipFree(rhovx_int));
	checkCuda(hipFree(rhovy_int));
	checkCuda(hipFree(rhovz_int));
	checkCuda(hipFree(Bx_int));
	checkCuda(hipFree(By_int));
	checkCuda(hipFree(Bz_int));
	checkCuda(hipFree(e_int));

	checkCuda(hipFree(grid_x));
	checkCuda(hipFree(grid_y));
	checkCuda(hipFree(grid_z));

	/* Free host data */
	// for (size_t ih = 0; ih < 8; ih++){
	// 	if (to_write_or_not[ih]){ // Don't forget to free the rest of the host buffers 
	// 		switch (ih)
	// 		{
	// 		case 0:
	// 			free(h_rho);
	// 			break;
	// 		case 1:
	// 			free(h_rhovx);
	// 			break;
	// 		case 2:
	// 			free(h_rhovy);
	// 			break;			
	// 		case 3:
	// 			free(h_rhovz);
	// 			break;			
	// 		case 4:
	// 			free(h_Bx);
	// 			break;			
	// 		case 5:
	// 			free(h_By);
	// 			break;			
	// 		case 6:
	// 			free(h_Bz);
	// 			break;			
	// 		case 7:
	// 			free(h_e);
	// 			break;			
	// 		default:
	// 			break;
	// 		}
	// 	}
	// }
	// free(to_write_or_not);
	return 0;
}