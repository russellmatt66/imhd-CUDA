/* 
WIP 
Test the Initial Conditions, and look at them
*/
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cufile.h>

#include "../include/initialize_od.cuh"


int main(int argc, char* argv[]){
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed, error: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }
    return 0;
}