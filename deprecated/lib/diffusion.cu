#include "hip/hip_runtime.h"
#include <stdio.h>
#include "diffusion.cuh"

#define IDX3D(i, j, k, Nx, Ny, Nz) ((k) * (Nx * Ny) + (i) * Ny + j) // parentheses are necessary to avoid calculating `i - 1 * Ny` or `k - 1 * (Nx * Ny)`

__device__ float numericalDiffusion(const int i, const int j, const int k, const float* fluid_var, 
    const float D, const float dx, const float dy, const float dz,
    const int ivf, const int Nx, const int Ny, const int Nz)
    {
        int cube_size = Nx * Ny * Nz;
        float num_diff = 0.0;

        // printf("(i, j, k) = (%d, %d, %d), Problem index is %d, cube size is %d, ivf is %d, computed linear index is %d, computed linear index should be %d, literal computed linear index is %d\n"
        //     , i, j, k, IDX3D(i, j, k-1, Nx, Ny, Nz) + ivf * cube_size, cube_size, ivf, IDX3D(i, j, k-1, Nx, Ny, Nz), (k - 1) * (Nx * Ny) + i * Ny + j, k - 1 * (Nx * Ny) + i * Ny + j);

        num_diff = D * (
            (1.0 / pow(dx, 2)) 
            * (
                fluid_var[IDX3D(i+1, j, k, Nx, Ny, Nz) + ivf * cube_size] 
                - 2.0*fluid_var[IDX3D(i, j, k, Nx, Ny, Nz) + ivf * cube_size] 
                + fluid_var[IDX3D(i-1, j, k, Nx, Ny, Nz) + ivf * cube_size]
                )
            + (1.0 / pow(dy, 2)) 
            * (
                fluid_var[IDX3D(i, j+1, k, Nx, Ny, Nz) + ivf * cube_size] 
                - 2.0*fluid_var[IDX3D(i, j, k, Nx, Ny, Nz) + ivf * cube_size] 
                + fluid_var[IDX3D(i, j-1, k, Nx, Ny, Nz) + ivf * cube_size]
                )
            + (1.0 / pow(dz, 2)) 
            * (
                fluid_var[IDX3D(i, j, k+1, Nx, Ny, Nz) + ivf * cube_size] 
                - 2.0*fluid_var[IDX3D(i, j, k, Nx, Ny, Nz) + ivf * cube_size] 
                + fluid_var[IDX3D(i, j, k-1, Nx, Ny, Nz) + ivf * cube_size]
                )
            );
        return num_diff;
    }

// Implements numerical diffusion on the front plane of the simulation grid (k = 0)
// Periodic boundary conditions are the reason
// 2nd-order central difference
// 19 AO, 9 MR
__device__ float numericalDiffusionFront(const int i, const int j, const float* fluid_var, 
    const float D, const float dx, const float dy, const float dz, 
    const int ivf, const int Nx, const int Ny, const int Nz)
    {   
        int cube_size = Nx * Ny * Nz;
        float num_diff = 0.0;
        num_diff = D * (
            (1.0 / pow(dx, 2)) 
                * (
                fluid_var[IDX3D(i+1, j, 0, Nx, Ny, Nz) + ivf * cube_size] 
                - 2.0*fluid_var[IDX3D(i, j, 0, Nx, Ny, Nz) + ivf * cube_size] 
                + fluid_var[IDX3D(i-1, j, 0, Nx, Ny, Nz) + ivf * cube_size]
                )
            + (1.0 / pow(dy, 2)) 
                * (
                fluid_var[IDX3D(i, j+1, 0, Nx, Ny, Nz) + ivf * cube_size] 
                - 2.0*fluid_var[IDX3D(i, j, 0, Nx, Ny, Nz) + ivf * cube_size] 
                + fluid_var[IDX3D(i, j-1, 0, Nx, Ny, Nz) + ivf * cube_size]
                )
            + (1.0 / pow(dz, 2)) 
                * (
                fluid_var[IDX3D(i, j, 1, Nx, Ny, Nz)  + ivf * cube_size] 
                - 2.0*fluid_var[IDX3D(i, j, 0, Nx, Ny, Nz) + ivf * cube_size] 
                + fluid_var[IDX3D(i, j, Nz - 2, Nx, Ny, Nz) + ivf * cube_size]
                )
            );
        return num_diff;
    }

// Implements numerical diffusion on the back plane of the simulation grid (k = Nz - 1)
// Periodic boundary conditions are the reason
// 2nd-order central difference
// 19 AO, 9 MR
__device__ float numericalDiffusionBack(const int i, const int j, const float* fluid_var, 
    const float D, const float dx, const float dy, const float dz, 
    const int ivf, const int Nx, const int Ny, const int Nz)
    {
        int cube_size = Nx * Ny * Nz;
        float num_diff = 0.0;
        num_diff = D * (
            (1.0 / pow(dx, 2)) 
            * (
                fluid_var[IDX3D(i+1, j, Nz - 1, Nx, Ny, Nz) + ivf * cube_size] 
                - 2.0*fluid_var[IDX3D(i, j, Nz - 1, Nx, Ny, Nz) + ivf * cube_size] 
                + fluid_var[IDX3D(i-1, j, Nz - 1, Nx, Ny, Nz) + ivf * cube_size]
                )
            + (1.0 / pow(dy, 2)) 
            * (
                fluid_var[IDX3D(i, j+1, Nz - 1, Nx, Ny, Nz) + ivf * cube_size] 
                - 2.0*fluid_var[IDX3D(i, j, Nz - 1, Nx, Ny, Nz) + ivf * cube_size] 
                + fluid_var[IDX3D(i, j-1, Nz - 1, Nx, Ny, Nz) + ivf * cube_size]
                )
            + (1.0 / pow(dz, 2)) 
            * (
                fluid_var[IDX3D(i, j, 1, Nx, Ny, Nz) + ivf * cube_size] 
                - 2.0*fluid_var[IDX3D(i, j, Nz - 1, Nx, Ny, Nz) + ivf * cube_size] 
                + fluid_var[IDX3D(i, j, Nz - 2, Nx, Ny, Nz) + ivf * cube_size])
            );
        return num_diff;
    }