#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "helper_functions.cuh"
#include "diffusion.cuh"
#include "kernels_od.cuh"
#include "kernels_od_fluxes.cuh"

#include <stdio.h>

#define IDX3D(i, j, k, Nx, Ny, Nz) (k) * (Nx * Ny) + (i) * (Ny) + j 

// Print values of intermediate variables
__global__ void PrintIntvar(const float* intvar, const float* fluidvar, const size_t Nx, const size_t Ny, const size_t Nz){
    int tidx = threadIdx.x + blockDim.x * blockIdx.x;
    int tidy = threadIdx.y + blockDim.y * blockIdx.y;
    int tidz = threadIdx.z + blockDim.z * blockIdx.z;

    int xthreads = blockDim.x * gridDim.x;
    int ythreads = blockDim.y * gridDim.y;
    int zthreads = blockDim.z * gridDim.z;

    int cube_size = Nx * Ny * Nz;

    for (int k = tidz; k < Nz; k += zthreads){
        for (int i = tidx; i < Nx; i += xthreads){
            for (int j = tidy; j < Ny; j += ythreads){
                float Bsq = B_sq(i, j, k, fluidvar, Nx, Ny, Nz);
                float ke = KE(i, j, k, fluidvar, Nx, Ny, Nz);
                float pressure = p(i, j, k, fluidvar, Bsq, ke, Nx, Ny, Nz);
                printf("Printing intvar values. "
                    "For (%d, %d, %d), the value of pressure is: %5.4f, Bsq: %5.4f, ke: %5.4f, " 
                    "int. rho: %5.4f, int. rhovx: %5.4f, int. rhovy: %5.4f, int. rhovz: %5.4f, " 
                    "int. Bx: %5.4f, int. By: %5.4f, int. Bz: %5.4f, int. energy: %5.4f, "
                    "gamma: %f\n", 
                    i, j, k, pressure, Bsq, ke, 
                    intvar[IDX3D(i, j, k, Nx, Ny, Nz)], intvar[IDX3D(i, j, k, Nx, Ny, Nz) +  cube_size], 
                    intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size], intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size],  
                    intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size], intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size],  
                    intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size], intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size], 
                    gamma);
            }
        }
    }
    return;
}

// Print values of fluid variables
__global__ void PrintFluidvar(const float* fluidvar, const size_t Nx, const size_t Ny, const size_t Nz){
    int tidx = threadIdx.x + blockDim.x * blockIdx.x;
    int tidy = threadIdx.y + blockDim.y * blockIdx.y;
    int tidz = threadIdx.z + blockDim.z * blockIdx.z;

    int xthreads = blockDim.x * gridDim.x;
    int ythreads = blockDim.y * gridDim.y;
    int zthreads = blockDim.z * gridDim.z;

    int cube_size = Nx * Ny * Nz;

    for (int k = tidz; k < Nz; k += zthreads){
        for (int i = tidx; i < Nx; i += xthreads){
            for (int j = tidy; j < Ny; j += ythreads){
                float Bsq = B_sq(i, j, k, fluidvar, Nx, Ny, Nz);
                float ke = KE(i, j, k, fluidvar, Nx, Ny, Nz);
                float pressure = p(i, j, k, fluidvar, Bsq, ke, Nx, Ny, Nz);
                printf("Printing fluidvar values. " 
                    "For (%d, %d, %d), the value of pressure is: %5.4f, Bsq: %5.4f, ke: %5.4f, " 
                    "rho: %5.4f, rhovx: %5.4f, rhovy: %5.4f, rhovz: %5.4f, " 
                    "Bx: %5.4f, By: %5.4f, Bz: %5.4f, energy: %5.4f, "
                    "gamma: %f\n", 
                    i, j, k, pressure, Bsq, ke, 
                    fluidvar[IDX3D(i, j, k, Nx, Ny, Nz)], fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) +  cube_size], 
                    fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size], fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size],  
                    fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size], fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size],  
                    fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size], fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size], 
                    gamma);
            }
        }
    }   
    return;
}