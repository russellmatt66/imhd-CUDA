#include "hip/hip_runtime.h"
#include <stdio.h>
#include <fcntl.h>
#include <hip/hip_runtime_api.h>
#include <cufile.h>

#include "gds.cuh"
#include "cufile_sample_utils.h"

#define IDX3D(i, j, k, Nx, Ny, Nz) (k * (Nx * Ny) + i * Nx + j)

void writeDataGDS(const char* filename, const float* data, const int size){
    int fd = -1;
    ssize_t ret = -1;

    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed, error: %s\n", hipGetErrorString(cudaStatus));
        return;
    }

    CUfileError_t status;
    CUfileDescr_t cf_descr;
    CUfileHandle_t cf_handle;

    status = cuFileDriverOpen();
    if (status.err != CU_FILE_SUCCESS){
        std::cerr << "cuFileDriverOpen failed, error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }

    const int bufferSize = sizeof(float) * size;

    fd = open(filename, O_CREAT | O_WRONLY, 0644);
	memset((void *)&cf_descr, 0, sizeof(CUfileDescr_t));
    cf_descr.handle.fd = open(filename, O_WRONLY);
    cf_descr.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;

    status = cuFileBufRegister(data, bufferSize, 0);
    if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "cuFile Buffer registration error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }

    status = cuFileHandleRegister(&cf_handle, &cf_descr);
        if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "cuFile Handle registration error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }

    ret = cuFileWrite(cf_handle, data, bufferSize, 0, 0);
    if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "cuFile File write error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }

    cuFileBufDeregister(data);
    cuFileHandleDeregister(cf_handle);

    status = cuFileDriverClose();
    if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "CUDA Driver close error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }
    std::cout << "Fluid data successfully written out" << std::endl;
    return;
}

// write x0 x1 ... xN-1 y0 y1 ... yN-1 z0 z1 ... zN-1
void writeGridBasisGDS(const char* filename, const float* x_grid, const float* y_grid, const float* z_grid, const int Nx, const int Ny, const int Nz){
    int fd = -1;
    ssize_t ret = -1;

    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed, error: %s\n", hipGetErrorString(cudaStatus));
        return;
    }

    CUfileError_t status;
    CUfileDescr_t cf_descr;
    CUfileHandle_t cf_handle;

    status = cuFileDriverOpen();
    if (status.err != CU_FILE_SUCCESS){
        std::cerr << "cuFileDriverOpen failed, error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }

    fd = open(filename, O_CREAT | O_WRONLY, 0644);
	memset((void *)&cf_descr, 0, sizeof(CUfileDescr_t));
    cf_descr.handle.fd = open(filename, O_WRONLY);
    cf_descr.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;

    status = cuFileHandleRegister(&cf_handle, &cf_descr);
        if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "cuFile Handle registration error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }

    int bufferSize_X = sizeof(float) * Nx; 
    int bufferSize_Y = sizeof(float) * Ny;
    int bufferSize_Z = sizeof(float) * Nz;

    status = cuFileBufRegister(x_grid, bufferSize_X, 0);
    if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "cuFile Buffer registration error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }

    ret = cuFileWrite(cf_handle, x_grid, bufferSize_X, 0, 0);
    if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "cuFile File write error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }
    
    status = cuFileBufRegister(y_grid, bufferSize_Y, 0);
    if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "cuFile Buffer registration error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }

    ret = cuFileWrite(cf_handle, y_grid, bufferSize_Y, 0, 0);
    if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "cuFile File write error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }


    status = cuFileBufRegister(z_grid, bufferSize_Z, 0);
    if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "cuFile Buffer registration error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }

    ret = cuFileWrite(cf_handle, z_grid, bufferSize_Z, 0, 0);
    if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "cuFile File write error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }

    cuFileBufDeregister(x_grid);
    cuFileBufDeregister(y_grid);
    cuFileBufDeregister(z_grid);
    cuFileHandleDeregister(cf_handle);

    status = cuFileDriverClose();
    if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "CUDA Driver close error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }
    return;
}

// Write x0 y0 z0 x0 y1 z0 x0 y2 z0 ... x1 y0 z0 x1 y1 z0 ...
void writeGridGDS(const char* filename, const float* x_grid, const float* y_grid, const float* z_grid, const int Nx, const int Ny, const int Nz){
    std::cout << "Writing grid data out" << std::endl;

    int fd = -1;
    ssize_t ret = -1;

    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed, error: %s\n", hipGetErrorString(cudaStatus));
        return;
    }

    CUfileError_t status;
    CUfileDescr_t cf_descr;
    CUfileHandle_t cf_handle;

    status = cuFileDriverOpen();
    if (status.err != CU_FILE_SUCCESS){
        std::cerr << "cuFileDriverOpen failed, error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }

    fd = open(filename, O_CREAT | O_WRONLY, 0644);
	memset((void *)&cf_descr, 0, sizeof(CUfileDescr_t));
    cf_descr.handle.fd = open(filename, O_WRONLY);
    cf_descr.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;

    status = cuFileHandleRegister(&cf_handle, &cf_descr);
        if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "cuFile Handle registration error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }

    // Create data buffer
    const int bufferSize = sizeof(float) * Nx * Ny * Nz * 3;
    float* data;
    // data = (float*)malloc(bufferSize);
    hipMalloc(&data, bufferSize);

    status = cuFileBufRegister(data, bufferSize, 0);
    if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "cuFile Buffer registration error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }

    // Write data
    /* SEGFAULT IN HERE: Data allocated to device is trying to be written by host */
    std::cout << "Creating buffer" << std::endl;
    for (int k = 0; k < Nz; k++){
        for (int i = 0; i < Nx; i++){
            for (int j = 0; j < Ny; j++){
                data[IDX3D(i, j, k, Nx, Ny, Nz)] = x_grid[i];
                data[IDX3D(i, j, k, Nx, Ny, Nz) + 1] = y_grid[j];
                data[IDX3D(i, j, k, Nx, Ny, Nz) + 2] = z_grid[k];
            }
        }
    }
    std::cout << "Buffer created" << std::endl; 

    ret = cuFileWrite(cf_handle, data, bufferSize, 0, 0);
    if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "cuFile File write error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }

    cuFileBufDeregister(data);
    cuFileHandleDeregister(cf_handle);

    status = cuFileDriverClose();
    if (status.err != CU_FILE_SUCCESS) {
        // fprintf(stderr, "cuFile error: %s\n", cuFileGetErrorString(status));
        std::cerr << "CUDA Driver close error: " << cuFileGetErrorString(status) << std::endl;
        return;
    }
    std::cout << "Grid data successfully written out" << std::endl;
    return;
}

__global__ void 

