#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include "kernels_od.cuh"
#include "kernels_od_fluxes.cuh"
#include "helper_functions.cuh"
#include "diffusion.cuh"

// row-major, column-minor order
#define IDX3D(i, j, k, Nx, Ny, Nz) ((k) * (Nx * Ny) + (i) * Ny + j) // parentheses are necessary to avoid calculating `i - 1 * Ny` or `k - 1 * (Nx * Ny)`

/* 
Needed to be refactored to compactify all data into three arrays:
(1) float *fluidvar
(2) float *intvar
(3) float *fluidvar_np1

Furthermore, there was a race condition amongst the intermediate variable calculation. 

To solve this, the intermediate variable's computation is moved outside of the fluid advance loop, and into its own kernel. 
They are precomputed after the initial conditions, and then compute in the loop while writing data out in order to add minimal synchronization barriers.   

Here is what the storage pattern looks like:
fluidvar -> [rho_{000}, rho_{010}, rho_{020}, ..., rho_{0,Ny-1,0}, rho_{100}, ..., rho_{Nx-1,Ny-1,Nz-1}, rhov_x_{000}, rhov_x_{010}, ... , e_{Nx-1,Ny-1,Nz-1}]
*/

// Global kernels
// 40 registers per thread
__global__ void SwapSimData(float* fluidvar, const float* fluidvar_np1, const int Nx, const int Ny, const int Nz)
    {
    int tidx = threadIdx.x + blockDim.x * blockIdx.x; 
    int tidy = threadIdx.y + blockDim.y * blockIdx.y;
    int tidz = threadIdx.z + blockDim.z * blockIdx.z;

    int xthreads = blockDim.x * gridDim.x;
    int ythreads = blockDim.y * gridDim.y;
    int zthreads = blockDim.z * gridDim.z;

    int cube_size = Nx * Ny * Nz;
    for (int k = tidz; k < Nz; k += zthreads){
        for (int i = tidx; i < Nx; i += xthreads){
            for (int j = tidy; j < Ny; j += ythreads){
                fluidvar[IDX3D(i, j, k, Nx, Ny, Nz)] = fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz)]; // rho
                fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + cube_size] = fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + cube_size]; // rhov_x
                fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size] = fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size]; // rhov_y
                fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size] = fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size]; // rhov_z
                fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size] = fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size]; // Bx
                fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size] = fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size]; // By 
                fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size] = fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size]; // Bz
                fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size] = fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size]; // e
            }
        }
    }
    return;
    }

// 66 registers per thread
__global__ void FluidAdvance(float* fluidvar_np1, const float* fluidvar, const float* intvar, 
    const float D, const float dt, const float dx, const float dy, const float dz, 
    const int Nx, const int Ny, const int Nz)
    {
    int tidx = threadIdx.x + blockDim.x * blockIdx.x; 
    int tidy = threadIdx.y + blockDim.y * blockIdx.y;
    int tidz = threadIdx.z + blockDim.z * blockIdx.z;

    int xthreads = blockDim.x * gridDim.x;
    int ythreads = blockDim.y * gridDim.y;
    int zthreads = blockDim.z * gridDim.z;

    int cube_size = Nx * Ny * Nz;

    for (int k = tidz + 1; k < Nz - 1; k += zthreads){
        for (int i = tidx + 1; i < Nx - 1; i += xthreads){
            for (int j = tidy + 1; j < Ny - 1; j += ythreads){
                fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz)] = LaxWendroffAdvRho(i, j, k, fluidvar, intvar, dt, dx, dy, dz, Nx, Ny, Nz) 
                                                            + numericalDiffusion(i, j, k, fluidvar, D, dx, dy, dz, 0, Nx, Ny, Nz); // rho
                fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + cube_size] =  LaxWendroffAdvRhoVX(i, j, k, fluidvar, intvar, dt, dx, dy, dz, Nx, Ny, Nz) 
                                                            + numericalDiffusion(i, j, k, fluidvar, D, dx, dy, dz, 1, Nx, Ny, Nz); // rhov_x
                fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size] =  LaxWendroffAdvRhoVY(i, j, k, fluidvar, intvar, dt, dx, dy, dz, Nx, Ny, Nz) 
                                                            + numericalDiffusion(i, j, k, fluidvar, D, dx, dy, dz, 2, Nx, Ny, Nz); // rhov_y
                fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size] =  LaxWendroffAdvRhoVZ(i, j, k, fluidvar, intvar, dt, dx, dy, dz, Nx, Ny, Nz) 
                                                            + numericalDiffusion(i, j, k, fluidvar, D, dx, dy, dz, 3, Nx, Ny, Nz); // rhov_z
                fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size] =  LaxWendroffAdvBX(i, j, k, fluidvar, intvar, dt, dx, dy, dz, Nx, Ny, Nz) 
                                                            + numericalDiffusion(i, j, k, fluidvar, D, dx, dy, dz, 4, Nx, Ny, Nz); // Bx
                fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size] =  LaxWendroffAdvBY(i, j, k, fluidvar, intvar, dt, dx, dy, dz, Nx, Ny, Nz) 
                                                            + numericalDiffusion(i, j, k, fluidvar, D, dx, dy, dz, 5, Nx, Ny, Nz); // By 
                fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size] =  LaxWendroffAdvBZ(i, j, k, fluidvar, intvar, dt, dx, dy, dz, Nx, Ny, Nz) 
                                                            + numericalDiffusion(i, j, k, fluidvar, D, dx, dy, dz, 6, Nx, Ny, Nz); // Bz
                fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size] =  LaxWendroffAdvE(i, j, k, fluidvar, intvar, dt, dx, dy, dz, Nx, Ny, Nz) 
                                                            + numericalDiffusion(i, j, k, fluidvar, D, dx, dy, dz, 7, Nx, Ny, Nz); // e
                
                // if (isnan(fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size])){
                //     float LW_val = LaxWendroffAdvRhoVZ(i, j, k, fluidvar, intvar, dt, dx, dy, dz, Nx, Ny, Nz);
                //     float dfn_val = numericalDiffusion(i, j, k, fluidvar, D, dx, dy, dz, 3, Nx, Ny, Nz);
                //     // printf("For (%d, %d, %d) the value of LWAdvance is %5.4f, and numericalDiffusion is %5.4f\n", i, j, k, LW_val, dfn_val);
                //     if (isnan(LW_val) && !isnan(dfn_val)){
                //         printf("LaxWendroffAdvRhoVZ is a problem, for (%d, %d, %d) the value of rhovz is %5.4f, intvar: %5.4f, XFRVZ: %5.4f, YFRVZ: %5.4f, ZFRVZ: %5.4f, XFRVZip1: %5.4f, YFRVZjp1: %5.4f, ZFRVZkp1: %5.4f\n", 
                //             i, j, k, fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size], intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size], 
                //             XFluxRhoVZ(i, j, k, intvar, Nx, Ny, Nz), YFluxRhoVZ(i, j, k, intvar, Nx, Ny, Nz), ZFluxRhoVZ(i, j, k, intvar, Nx, Ny, Nz), 
                //             XFluxRhoVZ(i + 1, j, k, intvar, Nx, Ny, Nz), YFluxRhoVZ(i, j + 1, k, intvar, Nx, Ny, Nz), ZFluxRhoVZ(i, j, k + 1, intvar, Nx, Ny, Nz));
                //     }
                // } 
            }
        }
    }
    return;
    }

// 120 registers per thread
__global__ void BoundaryConditions(volatile float* fluidvar_np1, const float* fluidvar, const float* intvar, 
    const float D, const float dt, const float dx, const float dy, const float dz,
    const int Nx, const int Ny, const int Nz)
    {
    int tidx = threadIdx.x + blockDim.x * blockIdx.x; 
    int tidy = threadIdx.y + blockDim.y * blockIdx.y;
    int tidz = threadIdx.z + blockDim.z * blockIdx.z;

    int xthreads = blockDim.x * gridDim.x;
    int ythreads = blockDim.y * gridDim.y;
    int zthreads = blockDim.z * gridDim.z;

    int cube_size = Nx * Ny * Nz;

    /* IMPLEMENT PBCs */
    // k = 0 and k = Nz - 1
    int k = 0;
    for (int i = tidx + 1; i < Nx - 1; i += xthreads){
        for (int j = tidy + 1; j < Ny - 1; j += ythreads){
            k = 0;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz)] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz)] + intvar[IDX3D(i, j, k, Nx, Ny, Nz)])
                                                        - 0.5 * (dt / dx) * (XFluxRho(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxRho(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxRho(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxRho(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxRho(i, j, k + 1, intvar, Nx, Ny, Nz) - ZFluxRho(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionFront(i, j, fluidvar, D, dx, dy, dz, 0, Nx, Ny, Nz);

            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + cube_size] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + cube_size])
                                                        - 0.5 * (dt / dx) * (XFluxRhoVX(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxRhoVX(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxRhoVX(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxRhoVX(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxRhoVX(i, j, k + 1, intvar, Nx, Ny, Nz) - ZFluxRhoVX(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionFront(i, j, fluidvar, D, dx, dy, dz, 1, Nx, Ny, Nz);
            
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size])
                                                        - 0.5 * (dt / dx) * (XFluxRhoVY(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxRhoVY(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxRhoVY(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxRhoVY(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxRhoVY(i, j, k + 1, intvar, Nx, Ny, Nz) - ZFluxRhoVY(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionFront(i, j, fluidvar, D, dx, dy, dz, 2, Nx, Ny, Nz);

            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size])
                                                        - 0.5 * (dt / dx) * (XFluxRhoVZ(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxRhoVZ(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxRhoVZ(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxRhoVZ(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxRhoVZ(i, j, k + 1, intvar, Nx, Ny, Nz) - ZFluxRhoVZ(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionFront(i, j, fluidvar, D, dx, dy, dz, 3, Nx, Ny, Nz);

            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size])
                                                        - 0.5 * (dt / dx) * (XFluxBX(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxBX(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxBX(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxBX(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxBX(i, j, k + 1, intvar, Nx, Ny, Nz) - ZFluxBX(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionFront(i, j, fluidvar, D, dx, dy, dz, 4, Nx, Ny, Nz);

            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size])
                                                        - 0.5 * (dt / dx) * (XFluxBY(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxBY(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxBY(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxBY(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxBY(i, j, k + 1, intvar, Nx, Ny, Nz) - ZFluxBY(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionFront(i, j, fluidvar, D, dx, dy, dz, 5, Nx, Ny, Nz);

            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size])
                                                        - 0.5 * (dt / dx) * (XFluxBZ(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxBZ(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxBZ(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxBZ(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxBZ(i, j, k + 1, intvar, Nx, Ny, Nz) - ZFluxBZ(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionFront(i, j, fluidvar, D, dx, dy, dz, 6, Nx, Ny, Nz);

            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size])
                                                        - 0.5 * (dt / dx) * (XFluxE(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxBZ(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxE(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxBZ(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxE(i, j, k + 1, intvar, Nx, Ny, Nz) - ZFluxBZ(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionFront(i, j, fluidvar, D, dx, dy, dz, 7, Nx, Ny, Nz);

            k = Nz - 1;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz)] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz)] + intvar[IDX3D(i, j, k, Nx, Ny, Nz)])
                                                        - 0.5 * (dt / dx) * (XFluxRho(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxRho(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxRho(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxRho(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxRho(i, j, 1, intvar, Nx, Ny, Nz) - ZFluxRho(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionBack(i, j, fluidvar, D, dx, dy, dz, 0, Nx, Ny, Nz);

            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + cube_size] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + cube_size])
                                                        - 0.5 * (dt / dx) * (XFluxRhoVX(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxRhoVX(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxRhoVX(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxRhoVX(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxRhoVX(i, j, 1, intvar, Nx, Ny, Nz) - ZFluxRhoVX(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionBack(i, j, fluidvar, D, dx, dy, dz, 1, Nx, Ny, Nz);
            
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size])
                                                        - 0.5 * (dt / dx) * (XFluxRhoVY(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxRhoVY(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxRhoVY(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxRhoVY(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxRhoVY(i, j, 1, intvar, Nx, Ny, Nz) - ZFluxRhoVY(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionBack(i, j, fluidvar, D, dx, dy, dz, 2, Nx, Ny, Nz);

            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size])
                                                        - 0.5 * (dt / dx) * (XFluxRhoVZ(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxRhoVZ(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxRhoVZ(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxRhoVZ(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxRhoVZ(i, j, 1, intvar, Nx, Ny, Nz) - ZFluxRhoVZ(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionBack(i, j, fluidvar, D, dx, dy, dz, 3, Nx, Ny, Nz);

            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size])
                                                        - 0.5 * (dt / dx) * (XFluxBX(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxBX(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxBX(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxBX(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxBX(i, j, 1, intvar, Nx, Ny, Nz) - ZFluxBX(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionBack(i, j, fluidvar, D, dx, dy, dz, 4, Nx, Ny, Nz);

            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size])
                                                        - 0.5 * (dt / dx) * (XFluxBY(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxBY(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxBY(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxBY(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxBY(i, j, 1, intvar, Nx, Ny, Nz) - ZFluxBY(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionBack(i, j, fluidvar, D, dx, dy, dz, 5, Nx, Ny, Nz);

            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size])
                                                        - 0.5 * (dt / dx) * (XFluxBZ(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxBZ(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxBZ(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxBZ(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxBZ(i, j, 1, intvar, Nx, Ny, Nz) - ZFluxBZ(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionBack(i, j, fluidvar, D, dx, dy, dz, 6, Nx, Ny, Nz);

            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size] = 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size])
                                                        - 0.5 * (dt / dx) * (XFluxE(i + 1, j, k, intvar, Nx, Ny, Nz) - XFluxBZ(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dy) * (YFluxE(i, j + 1, k, intvar, Nx, Ny, Nz) - YFluxBZ(i, j, k, intvar, Nx, Ny, Nz))
                                                        - 0.5 * (dt / dz) * (ZFluxE(i, j, 1, intvar, Nx, Ny, Nz) - ZFluxBZ(i, j, k, intvar, Nx, Ny, Nz));
                                                        + numericalDiffusionBack(i, j, fluidvar, D, dx, dy, dz, 7, Nx, Ny, Nz);
            
            // THEN, ACCUMULATE THE RESULTS ONTO ONE FACE, MAP AROUND TO THE OTHER, AND CONTINUE
            for (int ivf = 0; ivf < 8; ivf++){
                fluidvar_np1[IDX3D(i, j, 0, Nx, Ny, Nz) + ivf * cube_size] += fluidvar_np1[IDX3D(i, j, Nz - 1, Nx, Ny, Nz) + ivf * cube_size];
                fluidvar_np1[IDX3D(i, j, Nz - 1, Nx, Ny, Nz) + ivf * cube_size] = fluidvar_np1[IDX3D(i, j, 0, Nx, Ny, Nz) + ivf * cube_size];
            }
        }
    }

    /* 
    B.Cs on BOTTOM (II) 
    (i = 0, j, k) 
    and
    B.Cs on TOP (IV)
    (i = Nx-1, j, k) 
    */
    int i = 0; 
    for (int k = tidz; k < Nz; k += zthreads){ 
        for (int j = tidy; j < Ny; j += ythreads){
            i = 0;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz)] = 1.0; /* Magic vacuum number */
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + cube_size] = 0.0; // Rigid wall
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size] = 0.0;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size] = 0.0;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size] = 0.0; // Perfectly-conducting wall
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size] = 0.0; 
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size] = 0.0;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size] = p(i, j, k, fluidvar, 0.0, 0.0, Nx, Ny, Nz) / (gamma - 1.0);

            i = Nx - 1;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz)] = 1.0; /* Magic vacuum number */
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + cube_size] = 0.0; // Rigid wall
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size] = 0.0;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size] = 0.0;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size] = 0.0; // Perfectly-conducting wall
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size] = 0.0; 
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size] = 0.0;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size] = p(i, j, k, fluidvar, 0.0, 0.0, Nx, Ny, Nz) / (gamma - 1.0);
        }
    }

    /* 
    B.Cs on LEFT (V)
    (i, j = 0, k) 
    and
    B.Cs on RIGHT (III)
    (i, j = N-1, k) 
    */
    int j = 0;
    for (int k = tidz; k < Nz; k += zthreads){
        for (int i = tidx + 1; i < Nx - 1; i += xthreads){
            j = 0;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz)] = 1.0; /* Magic vacuum number */
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + cube_size] = 0.0; // Rigid wall
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size] = 0.0;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size] = 0.0;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size] = 0.0; // Perfectly-conducting wall
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size] = 0.0; 
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size] = 0.0;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size] = p(i, j, k, fluidvar, 0.0, 0.0, Nx, Ny, Nz) / (gamma - 1.0);
            
            j = Ny - 1;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz)] = 1.0; /* Magic vacuum number */
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + cube_size] = 0.0; // Rigid wall
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size] = 0.0;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size] = 0.0;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size] = 0.0; // Perfectly-conducting wall
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size] = 0.0; 
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size] = 0.0;
            fluidvar_np1[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size] = p(i, j, k, fluidvar, 0.0, 0.0, Nx, Ny, Nz) / (gamma - 1.0);
        }
    }

    return;
    }

// Device kernels
__device__ float LaxWendroffAdvRho(const int i, const int j, const int k, 
    const float* fluidvar, const float* intvar, 
    const float dt, const float dx, const float dy, const float dz,
    const int Nx, const int Ny, const int Nz)
    {
        return 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz)] + intvar[IDX3D(i, j, k, Nx, Ny, Nz)])
        - 0.5 * (dt / dx) * (XFluxRho(i+1, j, k, intvar, Nx, Ny, Nz) - XFluxRho(i, j, k, intvar, Nx, Ny, Nz))
        - 0.5 * (dt / dy) * (YFluxRho(i, j+1, k, intvar, Nx, Ny, Nz) - YFluxRho(i, j, k, intvar, Nx, Ny, Nz))
        - 0.5 * (dt / dz) * (ZFluxRho(i, j, k+1, intvar, Nx, Ny, Nz) - ZFluxRho(i, j, k, intvar, Nx, Ny, Nz));
    }

__device__ float LaxWendroffAdvRhoVX(const int i, const int j, const int k,
    const float* fluidvar, const float* intvar, 
    const float dt, const float dx, const float dy, const float dz,
    const int Nx, const int Ny, const int Nz)
    {
        int cube_size = Nx * Ny * Nz;
        return 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + cube_size])
                - 0.5 * (dt / dx) * (XFluxRhoVX(i+1, j, k, intvar, Nx, Ny, Nz) - XFluxRhoVX(i, j, k, intvar, Nx, Ny, Nz))
                - 0.5 * (dt / dy) * (YFluxRhoVX(i, j+1, k, intvar, Nx, Ny, Nz) - YFluxRhoVX(i, j, k, intvar, Nx, Ny, Nz))
                - 0.5 * (dt / dz) * (ZFluxRhoVX(i, j, k+1, intvar, Nx, Ny, Nz) - ZFluxRhoVX(i, j, k, intvar, Nx, Ny, Nz));  
    }

__device__ float LaxWendroffAdvRhoVY(const int i, const int j, const int k, 
    const float* fluidvar, const float* intvar,  
    const float dt, const float dx, const float dy, const float dz,
    const int Nx, const int Ny, const int Nz)
    {
        int cube_size = Nx * Ny * Nz;
        return 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 2 * cube_size])
                - 0.5 * (dt / dx) * (XFluxRhoVY(i+1, j, k, intvar, Nx, Ny, Nz) - XFluxRhoVY(i, j, k, intvar, Nx, Ny, Nz))
                - 0.5 * (dt / dy) * (YFluxRhoVY(i, j+1, k, intvar, Nx, Ny, Nz) - YFluxRhoVY(i, j, k, intvar, Nx, Ny, Nz))
                - 0.5 * (dt / dz) * (ZFluxRhoVY(i, j, k+1, intvar, Nx, Ny, Nz) - ZFluxRhoVY(i, j, k, intvar, Nx, Ny, Nz)); 
    }

__device__ float LaxWendroffAdvRhoVZ(const int i, const int j, const int k,
    const float* fluidvar, const float* intvar, 
    const float dt, const float dx, const float dy, const float dz,
    const int Nx, const int Ny, const int Nz)
    {
        int cube_size = Nx * Ny * Nz;
        return 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 3 * cube_size])
                - 0.5 * (dt / dx) * (XFluxRhoVZ(i+1, j, k, intvar, Nx, Ny, Nz) - XFluxRhoVZ(i, j, k, intvar, Nx, Ny, Nz))
                - 0.5 * (dt / dy) * (YFluxRhoVZ(i, j+1, k, intvar, Nx, Ny, Nz) - YFluxRhoVZ(i, j, k, intvar, Nx, Ny, Nz))
                - 0.5 * (dt / dz) * (ZFluxRhoVZ(i, j, k+1, intvar, Nx, Ny, Nz) - ZFluxRhoVZ(i, j, k, intvar, Nx, Ny, Nz)); 
    }

__device__ float LaxWendroffAdvBX(const int i, const int j, const int k,
    const float* fluidvar, const float* intvar, 
    const float dt, const float dx, const float dy, const float dz,
    const int Nx, const int Ny, const int Nz)
    {
        int cube_size = Nx * Ny * Nz;
        return 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 4 * cube_size])
                - 0.5 * (dt / dx) * (XFluxBX(i+1, j, k, intvar, Nx, Ny, Nz) - XFluxBX(i, j, k, intvar, Nx, Ny, Nz))
                - 0.5 * (dt / dy) * (YFluxBX(i, j+1, k, intvar, Nx, Ny, Nz) - YFluxBX(i, j, k, intvar, Nx, Ny, Nz))
                - 0.5 * (dt / dz) * (ZFluxBX(i, j, k+1, intvar, Nx, Ny, Nz) - ZFluxBX(i, j, k, intvar, Nx, Ny, Nz)); 
    }

__device__ float LaxWendroffAdvBY(const int i, const int j, const int k,
    const float* fluidvar, const float* intvar, 
    const float dt, const float dx, const float dy, const float dz,
    const int Nx, const int Ny, const int Nz)
    {
        int cube_size = Nx * Ny * Nz;
        return 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 5 * cube_size])
                - 0.5 * (dt / dx) * (XFluxBY(i+1, j, k, intvar, Nx, Ny, Nz) - XFluxBY(i, j, k, intvar, Nx, Ny, Nz))
                - 0.5 * (dt / dy) * (YFluxBY(i, j+1, k, intvar, Nx, Ny, Nz) - YFluxBY(i, j, k, intvar, Nx, Ny, Nz))
                - 0.5 * (dt / dz) * (ZFluxBY(i, j, k+1, intvar, Nx, Ny, Nz) - ZFluxBY(i, j, k, intvar, Nx, Ny, Nz)); 
    }

__device__ float LaxWendroffAdvBZ(const int i, const int j, const int k,
    const float* fluidvar, const float* intvar, 
    const float dt, const float dx, const float dy, const float dz,
    const int Nx, const int Ny, const int Nz)
    {
        int cube_size = Nx * Ny * Nz;
        return 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 6 * cube_size])
                - 0.5 * (dt / dx) * (XFluxBZ(i+1, j, k, intvar, Nx, Ny, Nz) - XFluxBZ(i, j, k, intvar, Nx, Ny, Nz))
                - 0.5 * (dt / dy) * (YFluxBZ(i, j+1, k, intvar, Nx, Ny, Nz) - YFluxBZ(i, j, k, intvar, Nx, Ny, Nz))
                - 0.5 * (dt / dz) * (ZFluxBZ(i, j, k+1, intvar, Nx, Ny, Nz) - ZFluxBZ(i, j, k, intvar, Nx, Ny, Nz)); 
    }

__device__ float LaxWendroffAdvE(const int i, const int j, const int k,
    const float* fluidvar, const float* intvar, 
    const float dt, const float dx, const float dy, const float dz,
    const int Nx, const int Ny, const int Nz)
    {
        int cube_size = Nx * Ny * Nz;
        return 0.5 * (fluidvar[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size] + intvar[IDX3D(i, j, k, Nx, Ny, Nz) + 7 * cube_size])
                - 0.5 * (dt / dx) * (XFluxE(i+1, j, k, intvar, Nx, Ny, Nz) - XFluxE(i, j, k, intvar, Nx, Ny, Nz))
                - 0.5 * (dt / dy) * (YFluxE(i, j+1, k, intvar, Nx, Ny, Nz) - YFluxE(i, j, k, intvar, Nx, Ny, Nz))
                - 0.5 * (dt / dz) * (ZFluxE(i, j, k+1, intvar, Nx, Ny, Nz) - ZFluxE(i, j, k, intvar, Nx, Ny, Nz)); 
    }